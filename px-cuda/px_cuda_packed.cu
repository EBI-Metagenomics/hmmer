#include "hip/hip_runtime.h"
#include <string.h>
#include "easel.h"
#include <x86intrin.h>
#include <math.h>
#include "esl_sse.h"
#include "esl_dsqdata.h"
#include "hmmer.h"
#include "px_cuda.h"
#include "hip/hip_runtime_api.h"


/*  HEY NICK!  The result checking code will only work correctly if HMMER is configured with --disable-avx
    and --disable-avx512 so that it uses SSE.  Otherwise, the RBV data in the oprofile will be striped for 
    256- or 512-bit vectors and badness will ensue */

#define KP 27  // number of characters in alphabet.  Make parameter.
#define MAX_BAND_WIDTH 10
#define NEGINFMASK 0x80808080
#define NUM_REPS 1000
#define MAX(a, b, c)\
  a = __vmaxs4(b, c);

 // asm("max.s32 %0, %1, %2;" : "=r"(a): "r"(b), "r"(c));


char * restripe_char(char *source, int source_chars_per_vector, int dest_chars_per_vector, int source_length, int *dest_length);
int *restripe_char_to_int(char *source, int source_chars_per_vector, int dest_ints_per_vector, int source_length, int *dest_length);


#define STEP_1()\
  sv0   = __vaddss4(sv0, *rsc);\
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, (last_row_fetched-row))) + offset;\
  xE0  = __vmaxs4(xE0, sv0);

#define STEP_2()\
  sv0   = __vaddss4(sv0, *rsc);\
  sv1   = __vaddss4(sv1, *(rsc+32));\
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, (last_row_fetched-row))) + offset;\
  xE0  = __vmaxs4(xE0, sv0);\
  xE0  = __vmaxs4(xE0, sv1);


#define STEP_3()\
  sv0   = __vaddss4(sv0, *rsc);\
  sv1   = __vaddss4(sv1, *(rsc+32));\
  sv2   = __vaddss4(sv2, *(rsc+64));\
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, (last_row_fetched-row))) + offset;\
  xE0  = __vmaxs4(xE0, sv0);\
  xE0  = __vmaxs4(xE0, sv1);\
  xE0  = __vmaxs4(xE0, sv2);

#define STEP_4()\
  sv0   = __vaddss4(sv0, *rsc);\
  sv1   = __vaddss4(sv1, *(rsc+32));\
  sv2   = __vaddss4(sv2, *(rsc+64));\
  sv3   = __vaddss4(sv3, *(rsc+96));\
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, (last_row_fetched-row))) + offset;\
  xE0  = __vmaxs4(xE0, sv0);\
  xE0  = __vmaxs4(xE0, sv1);\
  xE0  = __vmaxs4(xE0, sv2);\
  xE0  = __vmaxs4(xE0, sv3);

#define STEP_5()\
  sv0   = __vaddss4(sv0, *rsc);\
  sv1   = __vaddss4(sv1, *(rsc+32));\
  sv2   = __vaddss4(sv2, *(rsc+64));\
  sv3   = __vaddss4(sv3, *(rsc+96));\
  sv4   = __vaddss4(sv4, *(rsc+128));\
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, (last_row_fetched-row))) + offset;\
  xE0  = __vmaxs4(xE0, sv0);\
  xE0  = __vmaxs4(xE0, sv1);\
  xE0  = __vmaxs4(xE0, sv2);\
  xE0  = __vmaxs4(xE0, sv3);\
  xE0  = __vmaxs4(xE0, sv4);

#define STEP_6()\
  sv0   = __vaddss4(sv0, *rsc);\
  sv1   = __vaddss4(sv1, *(rsc+32));\
  sv2   = __vaddss4(sv2, *(rsc+64));\
  sv3   = __vaddss4(sv3, *(rsc+96));\
  sv4   = __vaddss4(sv4, *(rsc+128));\
  sv5   = __vaddss4(sv5, *(rsc+160));\
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, (last_row_fetched-row))) + offset;\
  xE0  = __vmaxs4(xE0, sv0);\
  xE0  = __vmaxs4(xE0, sv1);\
  xE0  = __vmaxs4(xE0, sv2);\
  xE0  = __vmaxs4(xE0, sv3);\
  xE0  = __vmaxs4(xE0, sv4);\
  xE0  = __vmaxs4(xE0, sv5);


#define STEP_7()\
  sv0   = __vaddss4(sv0, *rsc);\
  sv1   = __vaddss4(sv1, *(rsc+32));\
  sv2   = __vaddss4(sv2, *(rsc+64));\
  sv3   = __vaddss4(sv3, *(rsc+96));\
  sv4   = __vaddss4(sv4, *(rsc+128));\
  sv5   = __vaddss4(sv5, *(rsc+160));\
  sv6   = __vaddss4(sv6, *(rsc+192));\
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, (last_row_fetched-row))) + offset;\
  xE0  = __vmaxs4(xE0, sv0);\
  xE0  = __vmaxs4(xE0, sv1);\
  xE0  = __vmaxs4(xE0, sv2);\
  xE0  = __vmaxs4(xE0, sv3);\
  xE0  = __vmaxs4(xE0, sv4);\
  xE0  = __vmaxs4(xE0, sv5);\
  xE0  = __vmaxs4(xE0, sv6);

#define STEP_8()\
  sv0   = __vaddss4(sv0, *rsc);\
  sv1   = __vaddss4(sv1, *(rsc+32));\
  sv2   = __vaddss4(sv2, *(rsc+64));\
  sv3   = __vaddss4(sv3, *(rsc+96));\
  sv4   = __vaddss4(sv4, *(rsc+128));\
  sv5   = __vaddss4(sv5, *(rsc+160));\
  sv6   = __vaddss4(sv6, *(rsc+192));\
  sv7   = __vaddss4(sv7, *(rsc+224));\
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, (last_row_fetched-row))) + offset;\
  xE0  = __vmaxs4(xE0, sv0);\
  xE0  = __vmaxs4(xE0, sv1);\
  xE0  = __vmaxs4(xE0, sv2);\
  xE0  = __vmaxs4(xE0, sv3);\
  xE0  = __vmaxs4(xE0, sv4);\
  xE0  = __vmaxs4(xE0, sv5);\
  xE0  = __vmaxs4(xE0, sv6);\
  xE0  = __vmaxs4(xE0, sv7);

  #define STEP_9()\
  sv0   = __vaddss4(sv0, *rsc);\
  sv1   = __vaddss4(sv1, *(rsc+32));\
  sv2   = __vaddss4(sv2, *(rsc+64));\
  sv3   = __vaddss4(sv3, *(rsc+96));\
  sv4   = __vaddss4(sv4, *(rsc+128));\
  sv5   = __vaddss4(sv5, *(rsc+160));\
  sv6   = __vaddss4(sv6, *(rsc+192));\
  sv7   = __vaddss4(sv7, *(rsc+224));\
  sv8   = __vaddss4(sv8, *(rsc+256));\
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, (last_row_fetched-row))) + offset;\
  xE0  = __vmaxs4(xE0, sv0);\
  xE0  = __vmaxs4(xE0, sv1);\
  xE0  = __vmaxs4(xE0, sv2);\
  xE0  = __vmaxs4(xE0, sv3);\
  xE0  = __vmaxs4(xE0, sv4);\
  xE0  = __vmaxs4(xE0, sv5);\
  xE0  = __vmaxs4(xE0, sv6);\
  xE0  = __vmaxs4(xE0, sv7);\
  xE0  = __vmaxs4(xE0, sv8);


  #define STEP_10()\
  sv0   = __vaddss4(sv0, *rsc);\
  sv1   = __vaddss4(sv1, *(rsc+32));\
  sv2   = __vaddss4(sv2, *(rsc+64));\
  sv3   = __vaddss4(sv3, *(rsc+96));\
  sv4   = __vaddss4(sv4, *(rsc+128));\
  sv5   = __vaddss4(sv5, *(rsc+160));\
  sv6   = __vaddss4(sv6, *(rsc+192));\
  sv7   = __vaddss4(sv7, *(rsc+224));\
  sv8   = __vaddss4(sv8, *(rsc+256));\
  sv9   = __vaddss4(sv9, *(rsc+288));\
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, (last_row_fetched-row))) + offset;\
  xE0  = __vmaxs4(xE0, sv0);\
  xE0  = __vmaxs4(xE0, sv1);\
  xE0  = __vmaxs4(xE0, sv2);\
  xE0  = __vmaxs4(xE0, sv3);\
  xE0  = __vmaxs4(xE0, sv4);\
  xE0  = __vmaxs4(xE0, sv5);\
  xE0  = __vmaxs4(xE0, sv6);\
  xE0  = __vmaxs4(xE0, sv7);\
  xE0  = __vmaxs4(xE0, sv8);\
  xE0  = __vmaxs4(xE0, sv9);

#define ENSURE_DSQ(count)\
  if(row +count-1 >= last_row_fetched){\
    last_row_fetched = row + 31;\
    rsc_precompute = rbv[dsq[last_row_fetched - threadIdx.x]];\
  }

// Note that these CONVERT macros are different from the ones in the CPU SSV. They only implement the shifting necessary to prepare
// sv for the next row.  They don't include the STEP functionality.

#define CONVERT_1()\
  sv0 = __byte_perm(sv0, __shfl_up_sync(0xffffffff, sv0, 1), 0x2107);\
  if(threadIdx.x == 0){\
      sv0 = __byte_perm(sv0, 0x80, 0x3214);\
  }

#define CONVERT_2()\
  sv1 = __byte_perm(sv1, __shfl_up_sync(0xffffffff, sv1, 1), 0x2107);\
  if(threadIdx.x == 0){\
      sv1 = __byte_perm(sv1, 0x80, 0x3214);\
  }

#define CONVERT_3()\
  sv2 = __byte_perm(sv2, __shfl_up_sync(0xffffffff, sv2, 1), 0x2107);\
  if(threadIdx.x == 0){\
      sv2 = __byte_perm(sv2, 0x80, 0x3214);\
  }

#define CONVERT_4()\
  sv3 = __byte_perm(sv3, __shfl_up_sync(0xffffffff, sv3, 1), 0x2107);\
  if(threadIdx.x == 0){\
      sv3 = __byte_perm(sv3, 0x80, 0x3214);\
  }

#define CONVERT_5()\
  sv4 = __byte_perm(sv4, __shfl_up_sync(0xffffffff, sv4, 1), 0x2107);\
  if(threadIdx.x == 0){\
      sv4 = __byte_perm(sv4, 0x80, 0x3214);\
  }

#define CONVERT_6()\
  sv5 = __byte_perm(sv5, __shfl_up_sync(0xffffffff, sv5, 1), 0x2107);\
  if(threadIdx.x == 0){\
      sv5 = __byte_perm(sv5, 0x80, 0x3214);\
  }


#define CONVERT_7()\
  sv6 = __byte_perm(sv6, __shfl_up_sync(0xffffffff, sv6, 1), 0x2107);\
  if(threadIdx.x == 0){\
      sv6 = __byte_perm(sv6, 0x80, 0x3214);\
  }

#define CONVERT_8()\
  sv7 = __byte_perm(sv7, __shfl_up_sync(0xffffffff, sv7, 1), 0x2107);\
  if(threadIdx.x == 0){\
      sv7 = __byte_perm(sv7, 0x80, 0x3214);\
  }

#define CONVERT_9()\
    sv8 = __byte_perm(sv8, __shfl_up_sync(0xffffffff, sv8, 1), 0x2107);\
    if(threadIdx.x == 0){\
        sv8 = __byte_perm(sv8, 0x80, 0x3214);\
    }

#define CONVERT_10()\
    sv9 = __byte_perm(sv9, __shfl_up_sync(0xffffffff, sv9, 1), 0x2107);\
    if(threadIdx.x == 0){\
        sv9 = __byte_perm(sv9, 0x80, 0x3214);\
    }


__device__  uint calc_band_1(const __restrict__ uint8_t *dsq, int L, int Q, int q, int ** rbv){
  int sv0 = NEGINFMASK, xE0=NEGINFMASK, *rsc;
  int row=0, last_row_fetched = -1;
  int offset;
  int* rsc_precompute;
//printf("Starting calc_band_1, row = %d\n", row);
  offset = (q <<5)+threadIdx.x;
  ENSURE_DSQ(1)
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, 31)) + offset;
  row++;
  int num_iters = min(L, Q-(q +1)); // first band may start in middle of row
  while(row <= L-Q){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_1() 
      row++;
      offset+= 32;
      STEP_1() 
      row++;
      offset+= 32;
      STEP_1() 
      row++;
      offset+= 32;
      STEP_1()
      row++;
      num_iters -= 4;
    } 
    ENSURE_DSQ(num_iters)
    while(num_iters > 0){
      offset+= 32;
      STEP_1() 
      row++;
      num_iters--;
    }
      // at end of row, convert
      ENSURE_DSQ(1)
      offset = threadIdx.x;
      STEP_1()
      CONVERT_1()
      row++;
      num_iters = min(Q-1, L-row);
  }
  while(row <= L){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_1() 
      row++;
      offset+= 32;
      STEP_1() 
      row++;
      offset+= 32;
      STEP_1() 
      row++;
      offset+= 32;
      STEP_1()
      row++;
      num_iters -= 4;
    } 
    ENSURE_DSQ(num_iters)
    while(num_iters > 0){
      offset+= 32;
      STEP_1() 
      row++;
      num_iters--;
    }
    if(row <= L){
      // at end of row, convert
      ENSURE_DSQ(1)
      offset = threadIdx.x;
      STEP_1()
      CONVERT_1()
      row++;
      num_iters = min(Q-1, L-row);
    }
  }
  return xE0;   
}

__device__  uint calc_band_2(const __restrict__ uint8_t *dsq, int L, int Q, int q, int ** rbv){
  int sv0 = NEGINFMASK, xE0=NEGINFMASK, sv1 = NEGINFMASK, *rsc;
  int row=0, last_row_fetched = -1;
  int offset;
  int* rsc_precompute;
  offset = (q <<5)+threadIdx.x;
  ENSURE_DSQ(1)
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, 31)) + offset;
  row++;
  int num_iters = min(L, Q-(q +2)); // first band may start in middle of row
  while(row <= L-Q){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_2() 
      row++;
      offset+= 32;
      STEP_2() 
      row++;
      offset+= 32;
      STEP_2() 
      row++;
      offset+= 32;
      STEP_2()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters);
    while(num_iters > 0){
      offset+= 32;
      STEP_2() 
      row++;
      num_iters--;
    }
      // at end of row, convert
      ENSURE_DSQ(2)
      offset+=32;
      STEP_2()
      CONVERT_2()
      row++;
      offset = threadIdx.x;
      STEP_2()
      CONVERT_1()
      row++;
      num_iters = min(Q-2, L-row);
  }
  while(row <= L){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_2() 
      row++;
      offset+= 32;
      STEP_2() 
      row++;
      offset+= 32;
      STEP_2() 
      row++;
      offset+= 32;
      STEP_2()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters);
    while(num_iters > 0){
      offset+= 32;
      STEP_2() 
      row++;
      num_iters--;
    }
    if(row <= L){
      // at end of row, convert
      ENSURE_DSQ(2)
      offset += 32;
      STEP_2()
      CONVERT_2()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      STEP_2()
      row++;
    }
  }
  return xE0;   
}

__device__  uint calc_band_3(const __restrict__ uint8_t *dsq, int L, int Q, int q, int ** rbv){
  int sv0 = NEGINFMASK, xE0=NEGINFMASK, sv1 = NEGINFMASK, sv2 = NEGINFMASK, *rsc;
  int row=0, last_row_fetched = -1;
  int offset;
  int* rsc_precompute;
  offset = (q <<5)+threadIdx.x;
  ENSURE_DSQ(1)
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, 31)) + offset;
  row++;
  int num_iters = min(L, Q-(q +3)); // first band may start in middle of row
  while(row <= L-Q){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_3() 
      row++;
      offset+= 32;
      STEP_3() 
      row++;
      offset+= 32;
      STEP_3() 
      row++;
      offset+= 32;
      STEP_3()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_3() 
      row++;
      num_iters--;
    }
      // at end of row, convert
      ENSURE_DSQ(3)
      offset+=32;
      STEP_3()
      CONVERT_3()
      row++;  
      offset+=32;
      STEP_3()
      CONVERT_2()
      row++;
      offset = threadIdx.x;
      STEP_3()
      CONVERT_1()
      row++;
     // num_iters = Q-3;
      num_iters = min(Q-3, L-row);
  }
  num_iters = min(num_iters, L-row);
  while(row <= L){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_3() 
      row++;
      offset+= 32;
      STEP_3() 
      row++;
      offset+= 32;
      STEP_3() 
      row++;
      offset+= 32;
      STEP_3()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_3() 
      row++;
      num_iters--;
    }
   if(row <= L){
      // at end of row, convert
      ENSURE_DSQ(3)
      offset += 32;
      STEP_3()
      CONVERT_3()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_3()
      CONVERT_2()
      row++;
    }
    if(row <= L){
      //don't need to convert in last row
      STEP_3()
    }
  }
  return xE0;   
}


__device__  uint calc_band_4(const __restrict__ uint8_t *dsq, int L, int Q, int q, int ** rbv){
  int sv0 = NEGINFMASK, xE0=NEGINFMASK, sv1 = NEGINFMASK, sv2 = NEGINFMASK, sv3 = NEGINFMASK, *rsc;
  int row=0, last_row_fetched = -1;
  int offset;
  int* rsc_precompute;
  offset = (q <<5)+threadIdx.x;
  ENSURE_DSQ(1)
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, 31)) + offset;
  row++;
  int num_iters = min(L, Q-(q +4)); // first band may start in middle of row
  while(row <= L-Q){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_4() 
      row++;
      offset+= 32;
      STEP_4() 
      row++;
      offset+= 32;
      STEP_4() 
      row++;
      offset+= 32;
      STEP_4()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_4() 
      row++;
      num_iters--;
    }
      // at end of row, convert
      ENSURE_DSQ(4)
      offset+=32;
      STEP_4()
      CONVERT_4()
      row++;
      offset+=32;
      STEP_4()
      CONVERT_3()
      row++;  
      offset+=32;
      STEP_4()
      CONVERT_2()
      row++;
      offset = threadIdx.x;
      STEP_4()
      CONVERT_1()
      row++;
      num_iters = Q-4;
  }
  num_iters = min(num_iters, L-row);
  while(row <= L){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_4() 
      row++;
      offset+= 32;
      STEP_4() 
      row++;
      offset+= 32;
      STEP_4() 
      row++;
      offset+= 32;
      STEP_4()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_4() 
      row++;
      num_iters--;
    }
    if(row <= L){
      // at end of row, convert
      ENSURE_DSQ(4)
      offset += 32;
      STEP_4()
      CONVERT_4()
      row++;
    }
   if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_4()
      CONVERT_3()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_4()
      CONVERT_2()
      row++;
    }
    if(row <= L){
      //don't need to convert in last row
      STEP_4()
    }
  }

  return xE0;   
}

__device__  uint calc_band_5(const __restrict__ uint8_t *dsq, int L, int Q, int q, int ** rbv){
  int sv0 = NEGINFMASK, xE0=NEGINFMASK, sv1 = NEGINFMASK, sv2 = NEGINFMASK, sv3 = NEGINFMASK, sv4 = NEGINFMASK, *rsc;
  int row=0, last_row_fetched = -1;
  int offset;
  int* rsc_precompute;
  offset = (q <<5)+threadIdx.x;
  ENSURE_DSQ(1)
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, 31)) + offset;
  row++;
  int num_iters = min(L, Q-(q +5)); // first band may start in middle of row
  while(row <= L-Q){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_5() 
      row++;
      offset+= 32;
      STEP_5() 
      row++;
      offset+= 32;
      STEP_5() 
      row++;
      offset+= 32;
      STEP_5()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_5() 
      row++;
      num_iters--;
    }
      // at end of row, convert
      ENSURE_DSQ(5)
      offset+=32;
      STEP_5()
      CONVERT_5()
      row++;
      offset+=32;
      STEP_5()
      CONVERT_4()
      row++;  
      offset+=32;
      STEP_5()
      CONVERT_3()
      row++;
      offset+=32;
      STEP_5()
      CONVERT_2()
      row++;
      offset = threadIdx.x;
      STEP_5()
      CONVERT_1()
      row++;
      num_iters = Q-5;
  }
  num_iters = min(num_iters, L-row);
  while(row <= L){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_5() 
      row++;
      offset+= 32;
      STEP_5() 
      row++;
      offset+= 32;
      STEP_5() 
      row++;
      offset+= 32;
      STEP_5()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_5() 
      row++;
      num_iters--;
    }
    if(row <= L){
      // at end of row, convert
      ENSURE_DSQ(5)
      offset += 32;
      STEP_5()
      CONVERT_5()
      row++;
    }
   if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_5()
      CONVERT_4()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_5()
      CONVERT_3()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_5()
      CONVERT_2()
      row++;
    }
    if(row <= L){
      //don't need to convert in last row
      STEP_5()
    }
  }

  return xE0;   
}

__device__  uint calc_band_6(const __restrict__ uint8_t *dsq, int L, int Q, int q, int ** rbv){
  int sv0 = NEGINFMASK, xE0=NEGINFMASK, sv1 = NEGINFMASK, sv2 = NEGINFMASK, sv3 = NEGINFMASK, sv4 = NEGINFMASK, sv5 = NEGINFMASK, 
  *rsc;
  int row=0, last_row_fetched = -1;
  int offset;
  int* rsc_precompute;
  offset = (q <<5)+threadIdx.x;
  ENSURE_DSQ(1)
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, 31)) + offset;
  row++;
  int num_iters = min(L, Q-(q +6)); // first band may start in middle of row
  while(row <= L-Q){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_6() 
      row++;
      offset+= 32;
      STEP_6() 
      row++;
      offset+= 32;
      STEP_6() 
      row++;
      offset+= 32;
      STEP_6()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_6() 
      row++;
      num_iters--;
    }
      // at end of row, convert
      ENSURE_DSQ(6)
      offset+=32;
      STEP_6()
      CONVERT_6()
      row++;
      offset+=32;
      STEP_6()
      CONVERT_5()
      row++;
      offset+=32;
      STEP_6()
      CONVERT_4()
      row++;  
      offset+=32;
      STEP_6()
      CONVERT_3()
      row++;
      offset+=32;
      STEP_6()
      CONVERT_2()
      row++;
      offset = threadIdx.x;
      STEP_6()
      CONVERT_1()
      row++;
      num_iters = Q-6;
  }
  num_iters = min(num_iters, L-row);
  while(row <= L){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_6() 
      row++;
      offset+= 32;
      STEP_6() 
      row++;
      offset+= 32;
      STEP_6() 
      row++;
      offset+= 32;
      STEP_6()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_6() 
      row++;
      num_iters--;
    }
    if(row <= L){
      // at end of row, convert
      ENSURE_DSQ(6)
      offset += 32;
      STEP_6()
      CONVERT_6()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_6()
      CONVERT_5()
      row++;
    }  
   if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_6()
      CONVERT_4()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_6()
      CONVERT_3()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_6()
      CONVERT_2()
      row++;
    }
    if(row <= L){
      //don't need to convert in last row
      STEP_6()
    }
  }

  return xE0;   
}


__device__  uint calc_band_7(const __restrict__ uint8_t *dsq, int L, int Q, int q, int ** rbv){
  int sv0 = NEGINFMASK, xE0=NEGINFMASK, sv1 = NEGINFMASK, sv2 = NEGINFMASK, sv3 = NEGINFMASK, sv4 = NEGINFMASK, 
    sv5 = NEGINFMASK, sv6 =NEGINFMASK, *rsc;
  int row=0, last_row_fetched = -1;
  int offset;
  int* rsc_precompute;
  offset = (q <<5)+threadIdx.x;
  ENSURE_DSQ(1)
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, 31)) + offset;
  row++;
  int num_iters = min(L, Q-(q +7)); // first band may start in middle of row
  while(row <= L-Q){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_7() 
      row++;
      offset+= 32;
      STEP_7() 
      row++;
      offset+= 32;
      STEP_7() 
      row++;
      offset+= 32;
      STEP_7()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_7() 
      row++;
      num_iters--;
    }
      // at end of row, convert
      ENSURE_DSQ(7)
      offset+=32;
      STEP_7()
      CONVERT_7()
      row++;
      offset+=32;
      STEP_7()
      CONVERT_6()
      row++;
      offset+=32;
      STEP_7()
      CONVERT_5()
      row++;
      offset+=32;
      STEP_7()
      CONVERT_4()
      row++;  
      offset+=32;
      STEP_7()
      CONVERT_3()
      row++;
      offset+=32;
      STEP_7()
      CONVERT_2()
      row++;
      offset = threadIdx.x;
      STEP_7()
      CONVERT_1()
      row++;
      num_iters = Q-7;
  }
  num_iters = min(num_iters, L-row);
  while(row <= L){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_7() 
      row++;
      offset+= 32;
      STEP_7() 
      row++;
      offset+= 32;
      STEP_7() 
      row++;
      offset+= 32;
      STEP_7()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_7() 
      row++;
      num_iters--;
    }
    if(row <= L){
      // at end of row, convert
      ENSURE_DSQ(7)
      offset += 32;
      STEP_7()
      CONVERT_7()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_7()
      CONVERT_6()
      row++;
    }  
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_7()
      CONVERT_5()
      row++;
    }  
   if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_7()
      CONVERT_4()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_7()
      CONVERT_3()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_7()
      CONVERT_2()
      row++;
    }
    if(row <= L){
      //don't need to convert in last row
      STEP_7()
    }
  }

  return xE0;   
}

__device__  uint calc_band_8(const __restrict__ uint8_t *dsq, int L, int Q, int q, int ** rbv){
  int sv0 = NEGINFMASK, xE0=NEGINFMASK, sv1 = NEGINFMASK, sv2 = NEGINFMASK, sv3 = NEGINFMASK, sv4 = NEGINFMASK, 
    sv5 = NEGINFMASK, sv6 =NEGINFMASK, sv7= NEGINFMASK, *rsc;
  int row=0, last_row_fetched = -1;
  int offset;
  int* rsc_precompute;
  offset = (q <<5)+threadIdx.x;
  ENSURE_DSQ(1)
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, 31)) + offset;
  row++;
  int num_iters = min(L, Q-(q +8)); // first band may start in middle of row
  while(row <= L-Q){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_8() 
      row++;
      offset+= 32;
      STEP_8() 
      row++;
      offset+= 32;
      STEP_8() 
      row++;
      offset+= 32;
      STEP_8()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_8() 
      row++;
      num_iters--;
    }
      // at end of row, convert
      ENSURE_DSQ(8)
      offset+=32;
      STEP_8()
      CONVERT_8()
      row++;
      offset+=32;
      STEP_8()
      CONVERT_7()
      row++;
      offset+=32;
      STEP_8()
      CONVERT_6()
      row++;
      offset+=32;
      STEP_8()
      CONVERT_5()
      row++;
      offset+=32;
      STEP_8()
      CONVERT_4()
      row++;  
      offset+=32;
      STEP_8()
      CONVERT_3()
      row++;
      offset+=32;
      STEP_8()
      CONVERT_2()
      row++;
      offset = threadIdx.x;
      STEP_8()
      CONVERT_1()
      row++;
      num_iters = Q-8;
  }
  num_iters = min(num_iters, L-row);
  while(row <= L){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_8() 
      row++;
      offset+= 32;
      STEP_8() 
      row++;
      offset+= 32;
      STEP_8() 
      row++;
      offset+= 32;
      STEP_8()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_8() 
      row++;
      num_iters--;
    }
    if(row <= L){
      // at end of row, convert
      ENSURE_DSQ(8)
      offset += 32;
      STEP_8()
      CONVERT_8()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_8()
      CONVERT_7()
      row++;
    }  
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_8()
      CONVERT_6()
      row++;
    }  
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_8()
      CONVERT_5()
      row++;
    }  
   if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_8()
      CONVERT_4()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_8()
      CONVERT_3()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_8()
      CONVERT_2()
      row++;
    }
    if(row <= L){
      //don't need to convert in last row
      STEP_8()
    }
  }

  return xE0;   
}

__device__  uint calc_band_9(const __restrict__ uint8_t *dsq, int L, int Q, int q, int ** rbv){
  int sv0 = NEGINFMASK, xE0=NEGINFMASK, sv1 = NEGINFMASK, sv2 = NEGINFMASK, sv3 = NEGINFMASK, sv4 = NEGINFMASK, 
    sv5 = NEGINFMASK, sv6 =NEGINFMASK, sv7= NEGINFMASK, sv8 = NEGINFMASK, *rsc;
  int row=0, last_row_fetched = -1;
  int offset;
  int* rsc_precompute;
  offset = (q <<5)+threadIdx.x;
  ENSURE_DSQ(1)
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, 31)) + offset;
  row++;
  int num_iters = min(L, Q-(q +9)); // first band may start in middle of row
  while(row <= L-Q){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_9() 
      row++;
      offset+= 32;
      STEP_9() 
      row++;
      offset+= 32;
      STEP_9() 
      row++;
      offset+= 32;
      STEP_9()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_9() 
      row++;
      num_iters--;
    }
      // at end of row, convert
      ENSURE_DSQ(9)
      offset+=32;
      STEP_9()
      CONVERT_9()
      row++;
      offset+=32;
      STEP_9()
      CONVERT_8()
      row++;
      offset+=32;
      STEP_9()
      CONVERT_7()
      row++;
      offset+=32;
      STEP_9()
      CONVERT_6()
      row++;
      offset+=32;
      STEP_9()
      CONVERT_5()
      row++;
      offset+=32;
      STEP_9()
      CONVERT_4()
      row++;  
      offset+=32;
      STEP_9()
      CONVERT_3()
      row++;
      offset+=32;
      STEP_9()
      CONVERT_2()
      row++;
      offset = threadIdx.x;
      STEP_9()
      CONVERT_1()
      row++;
      num_iters = Q-9;
  }
  num_iters = min(num_iters, L-row);
  while(row <= L){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_9() 
      row++;
      offset+= 32;
      STEP_9() 
      row++;
      offset+= 32;
      STEP_9() 
      row++;
      offset+= 32;
      STEP_9()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_9() 
      row++;
      num_iters--;
    }
    if(row <= L){
      // at end of row, convert
      ENSURE_DSQ(9)
      offset += 32;
      STEP_9()
      CONVERT_9()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_9()
      CONVERT_8()
      row++;
    }  
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_9()
      CONVERT_7()
      row++;
    }  
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_9()
      CONVERT_6()
      row++;
    }  
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_9()
      CONVERT_5()
      row++;
    }  
   if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_9()
      CONVERT_4()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_9()
      CONVERT_3()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_9()
      CONVERT_2()
      row++;
    }
    if(row <= L){
      //don't need to convert in last row
      STEP_9()
    }
  }

  return xE0;   
}

__device__  uint calc_band_10(const __restrict__ uint8_t *dsq, int L, int Q, int q, int ** rbv){
  int sv0 = NEGINFMASK, xE0=NEGINFMASK,
  sv1 = NEGINFMASK, sv2 = NEGINFMASK, sv3 = NEGINFMASK, sv4 = NEGINFMASK, 
    sv5 = NEGINFMASK, sv6 =NEGINFMASK, sv7= NEGINFMASK, sv8 = NEGINFMASK, sv9=NEGINFMASK, *rsc;
  int row=0, last_row_fetched = -1;
  int offset;
  int* rsc_precompute;
  offset = (q <<5)+threadIdx.x;
  ENSURE_DSQ(1)
  rsc =  ((int *)__shfl_sync(0xffffffff, (uint64_t) rsc_precompute, 31)) + offset;
  row++;
  int num_iters = min(L, Q-(q +10)); // first band may start in middle of row
  while(row <= L-Q){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_10() 
      row++;
      offset+= 32;
      STEP_10() 
      row++;
      offset+= 32;
      STEP_10() 
      row++;
      offset+= 32;
      STEP_10()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_10() 
      row++;
      num_iters--;
    }
      // at end of row, convert
      ENSURE_DSQ(10)
      offset+=32;
      STEP_10()
      CONVERT_10()
      row++;
      offset+=32;
      STEP_10()
      CONVERT_9()
      row++;
      offset+=32;
      STEP_10()
      CONVERT_8()
      row++;
      offset+=32;
      STEP_10()
      CONVERT_7()
      row++;
      offset+=32;
      STEP_10()
      CONVERT_6()
      row++;
      num_iters = Q-10;
      offset+=32;
      STEP_10()
      CONVERT_5()
      row++;
      offset+=32;
      STEP_10()
      CONVERT_4()
      row++;  
      offset+=32;
      STEP_10()
      CONVERT_3()
      row++;
      offset+=32;
      STEP_10()
      CONVERT_2()
      row++;
      offset = threadIdx.x;
      STEP_10()
      CONVERT_1()
      row++;

  }
  num_iters = min(num_iters, L-row);
  while(row <= L){
    while (num_iters >= 4){
      ENSURE_DSQ(4)
      offset+= 32;
      STEP_10() 
      row++;
      offset+= 32;
      STEP_10() 
      row++;
      offset+= 32;
      STEP_10() 
      row++;
      offset+= 32;
      STEP_10()
      row++;
      num_iters -= 4;
    }
    ENSURE_DSQ(num_iters); 
    while(num_iters > 0){
      offset+= 32;
      STEP_10() 
      row++;
      num_iters--;
    }
    if(row <= L){
      // at end of row, convert
      ENSURE_DSQ(10)
      offset += 32;
      STEP_10()
      CONVERT_10()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_10()
      CONVERT_9()
      row++;
    }  
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_10()
      CONVERT_8()
      row++;
    }  
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_10()
      CONVERT_7()
      row++;
    }  
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_10()
      CONVERT_6()
      row++;
    }  
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_10()
      CONVERT_5()
      row++;
    }  
   if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_10()
      CONVERT_4()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_10()
      CONVERT_3()
      row++;
    }
    if(row <= L){
      // at end of row, convert
      offset += 32;
      STEP_10()
      CONVERT_2()
      row++;
    }
    if(row <= L){
      //don't need to convert in last row
      STEP_10()
    }
  }

  return xE0;   
}

__global__
void SSV_cuda(const __restrict__ uint8_t *dsq, int L, P7_OPROFILE *om, int8_t *retval){
  __shared__ uint4 shared_buffer[1024 *3];  //allocate one big lump that takes up all our shared memory
  int  Q = ((((om->M)-1) / (128)) + 1);
  int **rbv = (int **)shared_buffer; 

  // needs to scale w abc->Kp
  if(threadIdx.x < KP && threadIdx.y == 0 && threadIdx.z == 0){

    int rsc_length = (Q + MAX_BAND_WIDTH -1) * 128;  // 32 threads * 4 bytes 
    int cachable_rscs = ((48 *1024) - (((KP+1)/2)*2 * sizeof(uint *)))/rsc_length; // number of rbv entries that will fit in shared memory

   if(threadIdx.x < min(KP, cachable_rscs)){ //change 0 to KP to enable caching 
      rbv[threadIdx.x] = (int *)(rbv + ((KP+1)/2)*2) + (rsc_length/sizeof(int))*threadIdx.x;
      memcpy((void *) rbv[threadIdx.x], (void *) om->rbv[threadIdx.x], rsc_length);
    } 
    else{ 
      rbv[threadIdx.x]=(int *)(om->rbv[threadIdx.x]);
    }

  }
  __syncthreads();

  int xE = NEGINFMASK;

for(int num_reps = 0; num_reps < NUM_REPS; num_reps++){
  for (int i = 0; i < Q; i+=MAX_BAND_WIDTH) 
    {
    switch(min(MAX_BAND_WIDTH, Q-i)){
      case 1:
        xE = __vmaxs4(xE, calc_band_1(dsq, L, Q, i, rbv));
        break;
      case 2:
        xE = __vmaxs4(xE, calc_band_2(dsq, L, Q, i, rbv));
        break;
      case 3:
        xE = __vmaxs4(xE, calc_band_3(dsq, L, Q, i, rbv));
        break;
      case 4:
        xE = __vmaxs4(xE, calc_band_4(dsq, L, Q, i, rbv));
        break; 
      case 5:
        xE = __vmaxs4(xE, calc_band_5(dsq, L, Q, i, rbv));
        break; 
      case 6:
        xE = __vmaxs4(xE, calc_band_6(dsq, L, Q, i, rbv));
        break; 
      case 7:
        xE = __vmaxs4(xE, calc_band_7(dsq, L, Q, i, rbv));
        break; 
      case 8:
        xE = __vmaxs4(xE, calc_band_8(dsq, L, Q, i, rbv));
        break;  
      case 9:
        xE = __vmaxs4(xE, calc_band_9(dsq, L, Q, i, rbv));
        break; 
      case 10:
        xE = __vmaxs4(xE, calc_band_10(dsq, L, Q, i, rbv));
        break; 
      }
    }
  }

// Done with main loop.  Now reduce answer vector (xE) to one byte for return
  // Reduce 32 values to 16
  xE = __vmaxs4(xE, __shfl_down_sync(0x0000ffff, xE, 16));

 // Reduce 16 values to 8
  xE = __vmaxs4(xE, __shfl_down_sync(0x0000ff, xE, 8));
 
// Reduce 8 values to 4
  xE = __vmaxs4(xE, __shfl_down_sync(0x00000f, xE, 4));

// Reduce 4 values to 2
  xE = __vmaxs4(xE, __shfl_down_sync(0x000003, xE, 2));

// Reduce 2 values to 1
  xE = __vmaxs4(xE, __shfl_down_sync(0x000001, xE, 1));


  if((blockIdx.x == 0) &&(threadIdx.y ==0) && (threadIdx.x == 0)){ // only one thread writes result
  xE = __vmaxs4(xE, (xE>>16));
  xE = __vmaxs4(xE, (xE>>8)); 
    *retval = xE & 255; // low 8 bits of the word is the final result
  }
/*  if(threadIdx.x == 0){
    free(dsq_parsed);
  }   */  
  return; 
}  



// GPU kernel that copies values from the CPU version of an OPROFILE to one on the GPU.  Should generally only be called on one GPU core
__global__ void copy_oprofile_values_to_card(P7_OPROFILE *the_profile, float tauBM, float scale_b, float scale_w, int16_t base_w, int16_t ddbound_w, int L, int M, int V, int max_length, int allocM, int allocQb, int allocQw, int allocQf, int mode, float nj, int is_shadow, int8_t **rbv){

  the_profile->tauBM = tauBM;
  the_profile->scale_b = scale_b;
  the_profile->scale_w = scale_w;
  the_profile->base_w = base_w;
  the_profile->ddbound_w = ddbound_w;
  the_profile->L = L;
  the_profile->M = M;
  the_profile->V = V;
  the_profile->max_length = max_length;
  the_profile->allocM = allocM;
  the_profile->allocQb = allocQb;
  the_profile->allocQw = allocQw;
  the_profile->allocQf = allocQf;
  the_profile->mode = mode;
  the_profile->nj = nj;
  the_profile->is_shadow = is_shadow;
  the_profile->rbv = rbv;
}


// GPU kernel that initializes a filtermx structure
__global__ void initialize_filtermx_on_card(P7_FILTERMX *the_filtermx){
  the_filtermx->M = 0;
  the_filtermx->Vw = 64; // 32 cores * 32 bits = 1024 bits = 128 bytes = 64 * 16 bits
  the_filtermx->allocM = 0;
  the_filtermx->dp = NULL;
  the_filtermx->type = p7F_SSVFILTER;
}


// allocates and populates a P7_OPROFILE structure on a CUDA card that matches the one passed as its argument
P7_OPROFILE *create_oprofile_on_card(P7_OPROFILE *the_profile){
  P7_OPROFILE *cuda_OPROFILE;
  hipError_t err;
  int Q = P7_Q(the_profile->M, the_profile->V);

  if(hipMalloc(&cuda_OPROFILE, sizeof(P7_OPROFILE)) != hipSuccess){

    err = hipGetLastError();
    printf("Error: %s\n", hipGetErrorString(err));
    p7_Fail((char *) "Unable to allocate memory in create_oprofile_on_card");
  }

  // allocate and copy over rbv 2-D array
  unsigned int **cuda_rbv;
  if(hipMalloc(&cuda_rbv, the_profile->abc->Kp * sizeof(unsigned int *)) != hipSuccess){
    p7_Fail((char *) "Unable to allocate memory in create_oprofile_on_card");
  }
  int i;
  char *restriped_rbv;
  int restriped_rbv_size;

  unsigned int **cuda_rbv_temp = cuda_rbv; // use this variable to copy rbv pointers into CUDA array 
  for(i = 0; i < the_profile->abc->Kp; i++){
    int *cuda_rbv_entry;
  restriped_rbv = restripe_char ((char*)(the_profile->rbv[i]), the_profile->V, 128, Q * the_profile->V, &restriped_rbv_size);
  //restriped_rbv = (int *) restripe_char((char *)(the_profile->rbv[i]), the_profile->V, 128, Q * the_profile->V, &restriped_rbv_size);

    if(hipMalloc(&cuda_rbv_entry, restriped_rbv_size) != hipSuccess){
      p7_Fail((char *) "Unable to allocate memory in create_oprofile_on_card");
    }

    if(hipMemcpy(cuda_rbv_entry, restriped_rbv, restriped_rbv_size, hipMemcpyHostToDevice) != hipSuccess){
      p7_Fail((char *) "Unable to copy data in create_oprofile_on_card");
    }

    if(hipMemcpy(cuda_rbv_temp, &cuda_rbv_entry, sizeof(int *) , hipMemcpyHostToDevice) != hipSuccess){
      p7_Fail((char *) "Unable to copy data in create_oprofile_on_card");
    }
    cuda_rbv_temp +=1;
  }
 

  // copy over base parameters.  Only call this kernel on one core because it just assigns values to fields in the data structure and has no parallelism
  copy_oprofile_values_to_card<<<1,1>>>(cuda_OPROFILE, the_profile->tauBM, the_profile->scale_b, the_profile->scale_w, the_profile->base_w, the_profile->ddbound_w, the_profile->L, the_profile->M, the_profile->V, the_profile->max_length, the_profile->allocM, the_profile->allocQb, the_profile->allocQw, the_profile->allocQf, the_profile->mode, the_profile->nj, the_profile->is_shadow, (int8_t **) cuda_rbv);

 return cuda_OPROFILE;
}

void destroy_oprofile_on_card(P7_OPROFILE *cpu_oprofile, P7_OPROFILE *cuda_oprofile){
  int i;
  for(i = 0; i < cpu_oprofile->abc->Kp; i++){
    hipFree(cuda_oprofile->rbv[i]);
  }
  hipFree(cuda_oprofile->rbv);
  hipFree(cuda_oprofile);
}

P7_FILTERMX *create_filtermx_on_card(){
  P7_FILTERMX *the_filtermx;
  
  if(hipMalloc(&the_filtermx, sizeof(P7_FILTERMX)) != hipSuccess){
    p7_Fail((char *) "Unable to allocate memory in create_filtermx_on_card");
  }
  initialize_filtermx_on_card<<<1,1>>>(the_filtermx);
  return the_filtermx;
}


char * restripe_char(char *source, int source_chars_per_vector, int dest_chars_per_vector, int source_length, int *dest_length){
  char *dest;
  int dest_num_vectors, unpadded_dest_vectors;
  int source_num_vectors;
  source_num_vectors = source_length/source_chars_per_vector;
  if(source_num_vectors * source_chars_per_vector != source_length){
    source_num_vectors++;
  }
  unpadded_dest_vectors = source_length/dest_chars_per_vector;
  if(unpadded_dest_vectors * dest_chars_per_vector != source_length){
    unpadded_dest_vectors++;  //round up if source length isn't a multiple of the dest vector length
  }
 // printf("Unpadded_dest_vectors = %d. ", unpadded_dest_vectors);
  dest_num_vectors = unpadded_dest_vectors + MAX_BAND_WIDTH -1; // add extra vectors for SSV wrap-around

  dest = (char *) malloc(dest_num_vectors * dest_chars_per_vector);
  *dest_length = dest_num_vectors * dest_chars_per_vector;
  int source_pos, dest_pos;
  int i;

  for(i = 0; i < source_length; i++){
    source_pos = ((i % source_num_vectors) * source_chars_per_vector) + (i / source_num_vectors);
    dest_pos = ((i % unpadded_dest_vectors) * dest_chars_per_vector) + (i / unpadded_dest_vectors);
    dest[dest_pos] = (int) source[source_pos];
  }

  // pad out the dest vector with zeroes if necessary
  for(; i < unpadded_dest_vectors * dest_chars_per_vector; i++){
      dest_pos = ((i % unpadded_dest_vectors) * dest_chars_per_vector) + (i / unpadded_dest_vectors);
    //  printf("Padding 0 at location %d \n", dest_pos);
    dest[dest_pos] = -128;
  }

  // add the extra copies of the early vectors to support the SSV wrap-around
  for(int source = 0; i < dest_num_vectors * dest_chars_per_vector; i++){
    dest[i] = dest[source];
   // printf("Padding from location %d to location %d\n", source, i);
    source++;
  }

  return dest;

}


int *restripe_char_to_int(char *source, int source_chars_per_vector, int dest_ints_per_vector, int source_length, int *dest_length){
  int *dest;
  int dest_num_vectors, source_num_vectors, unpadded_dest_vectors;

  source_num_vectors = source_length/source_chars_per_vector;
  if(source_num_vectors * source_chars_per_vector != source_length){
    source_num_vectors++;
  }
  unpadded_dest_vectors = source_length/dest_ints_per_vector;
  if(unpadded_dest_vectors * dest_ints_per_vector != source_length){
    unpadded_dest_vectors++;  //round up if source length isn't a multiple of the dest vector length
  }
 // printf("Unpadded_dest_vectors = %d. ", unpadded_dest_vectors);
  dest_num_vectors = unpadded_dest_vectors + MAX_BAND_WIDTH -1; // add extra vectors for SSV wrap-around
  dest = (int *) malloc(dest_num_vectors * dest_ints_per_vector * sizeof(int));
  *dest_length = dest_num_vectors * dest_ints_per_vector *sizeof(int);
  //printf("Padded dest_num_vectors = %d. Dest_length = %d\n", dest_num_vectors, *dest_length);

  int source_pos, dest_pos;
  int i;

  for(i = 0; i < source_length; i++){
    source_pos = ((i % source_num_vectors) * source_chars_per_vector) + (i / source_num_vectors);
    dest_pos = ((i % unpadded_dest_vectors) * dest_ints_per_vector) + (i / unpadded_dest_vectors);
    dest[dest_pos] = (int) source[source_pos];
  }

  // pad out the dest vector with zeroes if necessary
  for(; i < unpadded_dest_vectors * dest_ints_per_vector; i++){
      dest_pos = ((i % unpadded_dest_vectors) * dest_ints_per_vector) + (i / unpadded_dest_vectors);
    //  printf("Padding 0 at location %d \n", dest_pos);
    dest[dest_pos] = -128;
  }

  // add the extra copies of the early vectors to support the SSV wrap-around
  for(int source = 0; i < dest_num_vectors * dest_ints_per_vector; i++){
    dest[i] = dest[source];
   // printf("Padding from location %d to location %d\n", source, i);
    source++;
  }

  return dest;
}


int
p7_SSVFilter_shell_sse(const ESL_DSQ *dsq, int L, const __restrict__
  P7_OPROFILE *om, P7_FILTERMX *fx, float *ret_sc, P7_OPROFILE *card_OPROFILE, P7_FILTERMX *card_FILTERMX, int num)
{
  int      Q          = P7_Q(om->M, p7_VWIDTH_SSE);
  __m128i  hv         = _mm_set1_epi8(-128);
  __m128i  neginfmask = _mm_insert_epi8( _mm_setzero_si128(), -128, 0);
  __m128i *dp;
  __m128i *rbv;
  __m128i  mpv;
  __m128i  sv;
  int8_t   h, *card_h;
  int      i,q;
  int      status;

  //printf("Dim %d %d\n", ((((om->M)-1) / (128)) + 1) * 32, L);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds, seconds, gcups;
  //char *card_rbv= NULL;
  uint8_t *card_dsq;
  int warps_per_block;
  dim3 threads_per_block, num_blocks;
  hipError_t err;
  if (( status = p7_filtermx_Reinit(fx, om->M) ) != eslOK) goto FAILURE;
  fx->M    = om->M;
  fx->Vw   = p7_VWIDTH_SSE / sizeof(int16_t); // A hack. FILTERMX wants Vw in units of int16_t. 
  fx->type = p7F_SSVFILTER;
  dp       = (__m128i *) fx->dp;

  hipMalloc((void **) &card_h, 1);
  err = hipGetLastError();
  hipMalloc((void**)  &card_dsq, L+31);  //Pad out so that we can grab dsq four bytes at a time
  hipMemset((void *) card_dsq, 0, L+31);
  hipMemcpy(card_dsq, (dsq+ 1), L, hipMemcpyHostToDevice);

  hipEventRecord(start);
  num_blocks.x = 20;
  num_blocks.y = 1;
  num_blocks.z = 1;
  warps_per_block = 32;
  threads_per_block.x = 32;
  threads_per_block.y = warps_per_block;
  threads_per_block.z = 1;
  
  //uint *check_array, *check_array_cuda;
  /*check_array = (uint *) calloc(L * ((((om->M)-1) / (32)) + 1) * 32 * sizeof(uint), 1);
  hipMalloc((void**)  &check_array_cuda,L * ((((om->M)-1) / (32)) + 1) * 32 * sizeof(uint)); 
  hipMemcpy(check_array_cuda, check_array, L * ((((om->M)-1) / (32)) + 1) * 32* sizeof(uint), hipMemcpyHostToDevice);
 */

  SSV_cuda <<<num_blocks, threads_per_block>>>(card_dsq, L, card_OPROFILE, card_h);
  int8_t h_compare;
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  seconds = milliseconds/1000;
  hipDeviceSynchronize();
  hipMemcpy(&h_compare, card_h, 1, hipMemcpyDeviceToHost);
  gcups = ((((float) (om->M * L) *(float) NUM_REPS)/seconds)/1e9) * (float)(num_blocks.x * num_blocks.y *num_blocks.z) * (float)warps_per_block;
  //printf("M = %d, L = %d, seconds = %f, GCUPS = %f\n", om->M, L, seconds, gcups); 
 // printf("length = %d, M = %d, gcups =%f\n", L, om->M, gcups);
 //printf("%f\n", gcups);

  err = hipGetLastError();
  if(err != hipSuccess){
    printf("Error: %s\n", hipGetErrorString(err));
  }
  // Compare CUDA result and SSE
  //hipMemcpy(check_array, check_array_cuda, L * ((((om->M)-1) / (32)) + 1) * 32* sizeof(uint), hipMemcpyDeviceToHost);
 // card_Q = ((((om->M)-1) / (128)) + 1);
  mpv = hv;
  for (q = 0; q < Q; q++)
    dp[q] = hv;

  for (i = 1; i <= L; i++)
    {
      rbv = (__m128i *) om->rbv[dsq[i]];
      
      for (q = 0; q < Q; q++)
        {
          sv    = _mm_adds_epi8(mpv, rbv[q]);
          hv    = _mm_max_epi8(hv, sv);
          mpv   = dp[q];
          dp[q] = sv;
        }  
      mpv = esl_sse_rightshift_int8(sv, neginfmask);
    // Check row against GPU
   /*   for(int elem= 0; elem < Q * p7_VWIDTH_SSE; elem++){
        int card_index = ((i -1) * card_Q * 128) + ((elem % card_Q) * 128) + (elem/card_Q);
        int cpu_index =  ((elem % Q) * p7_VWIDTH_SSE) + (elem/Q);
        uint8_t card_val = ((uint8_t *)check_array)[card_index];
        uint8_t cpu_val = ((uint8_t *)dp)[cpu_index];
        if(card_val != cpu_val){
          printf("Row value miss-match at row %d, position %d. CPU had %d, GPU had %d.  CPU index was %d, GPU index was %d\n", (i-1), elem, cpu_val, card_val, cpu_index, (card_index - ((i -1) * card_Q * 128)));
        }
      } */
     int cuda_length =0;
      char *cuda_row = restripe_char((char *) dp, 16, 128, Q * 16, &cuda_length);
  /*    printf("CPU %04d ", i-1);
      for(int v = 0; v < cuda_length/4; v++){
        printf("%08x ", ((uint32_t *) cuda_row)[v]);
      }
      printf("\n"); */ 
    } 
  h = esl_sse_hmax_epi8(hv);
  hipFree(card_h);
  hipFree(card_dsq);

  for(i = 0; i < om->abc->Kp; i++){

  }
  printf("score %x \n", h_compare);
  if(h != h_compare){
    printf("Final result miss-match: %x (CUDA) vs %x (CPU) on sequence %d with length %d\n\n", h_compare, h, num, L);
  } 
  if(om->V != 16){
    printf("Ignore any result miss-matches, as HMMER was not compiled to use only SSE instructions\n");
  }
 float known_good;  
 
 p7_SSVFilter_base_sse(dsq, L, om, fx, &known_good);
  if (h == 127)  
    { *ret_sc = eslINFINITY;
      if(*ret_sc != known_good){
        printf("miss-match with known good result %f vs %f\n", *ret_sc, known_good);
      }                  
      return eslOK;
     }
  else if (h > -128)
    { 
      *ret_sc = ((float) h + 128.) / om->scale_b + om->tauBM - 2.0;   // 2.0 is the tauNN/tauCC "2 nat approximation"
      *ret_sc += 2.0 * logf(2.0 / (float) (L + 2)); 
      if(*ret_sc != known_good){
        printf("miss-match with known good result %f vs %f\n", *ret_sc, known_good);
      }                  
      return eslOK;
    }
  else 
    {
      *ret_sc = -eslINFINITY;
            if(*ret_sc != known_good){
        printf("miss-match with known good result %f vs %f\n", *ret_sc, known_good);
      }     
      return eslOK;
    }
    
 FAILURE:
  *ret_sc = -eslINFINITY;
        if(*ret_sc != known_good){
        printf("miss-match with known good result %f vs %f\n", *ret_sc, known_good);
      }     
  return status;
}


static ESL_OPTIONS options[] = {
  /* name           type      default  env  range  toggles reqs incomp  help                               docgroup*/
  { (char *) "-h",        eslARG_NONE,  FALSE,  NULL, NULL,   NULL,  NULL, NULL,  (char *) "show brief help on version and usage",  0 },
  {  (char *) "-s",        eslARG_INT,      (char *) "0",  NULL, NULL,   NULL,  NULL, NULL,  (char *) "set random number seed to <n>",         0 },
  {  0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
};
static char usage[]  = "[-options] <hmmfile> <seqfile>";
static char banner[] = "px, the first parallel tests of H4";

int
main(int argc, char **argv)
{
  ESL_GETOPTS    *go      = p7_CreateDefaultApp(options, 2, argc, argv, banner, usage);
  char           *hmmfile = esl_opt_GetArg(go, 1);
  char           *seqfile = esl_opt_GetArg(go, 2);
  ESL_ALPHABET   *abc     = NULL;
  P7_HMMFILE     *hfp     = NULL;
  P7_BG          *bg      = NULL;
  P7_HMM         *hmm     = NULL;
  P7_PROFILE     *gm      = NULL;
  P7_OPROFILE    *om      = NULL;
  ESL_DSQDATA    *dd      = NULL;
  P7_ENGINE      *eng     = NULL;
  ESL_DSQDATA_CHUNK *chu = NULL;
  int             ncore   = 1;
  int  i;
  int             status;

  /* Read in one HMM */
  if (p7_hmmfile_OpenE(hmmfile, NULL, &hfp, NULL) != eslOK) p7_Fail( (char *) "Failed to open HMM file %s", hmmfile);
  if (p7_hmmfile_Read(hfp, &abc, &hmm)            != eslOK) p7_Fail( (char *) "Failed to read HMM");
  
  /* Configure a profile from the HMM */
  bg = p7_bg_Create(abc);
  gm = p7_profile_Create (hmm->M, abc);
  om = p7_oprofile_Create(hmm->M, abc);
  p7_profile_Config   (gm, hmm, bg);
  p7_oprofile_Convert (gm, om);
  P7_OPROFILE *card_OPROFILE;
  card_OPROFILE = create_oprofile_on_card((P7_OPROFILE *) om);
  hipDeviceSynchronize();
  P7_FILTERMX *card_FILTERMX;
  card_FILTERMX = create_filtermx_on_card();
  hipDeviceSynchronize();
  p7_bg_SetFilter(bg, om->M, om->compo);

  //uint64_t sequence_id = 0;
  uint64_t num_hits = 0;
  int count;
  hipGetDeviceCount(&count);
  //printf("Found %d CUDA devices\n", count);
  hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024*1024*16);
  /* Open sequence database */
  status = esl_dsqdata_Open(&abc, seqfile, ncore, &dd);
  if      (status == eslENOTFOUND) p7_Fail( (char *) "Failed to open dsqdata files:\n  %s",    dd->errbuf);
  else if (status == eslEFORMAT)   p7_Fail( (char *) "Format problem in dsqdata files:\n  %s", dd->errbuf);
  else if (status != eslOK)        p7_Fail( (char *) "Unexpected error in opening dsqdata (code %d)", status);

  eng = p7_engine_Create(abc, NULL, NULL, gm->M, 400);

  while (( status = esl_dsqdata_Read(dd, &chu)) == eslOK)  
    {
      for (i = 1; i < chu->N; i++)
	{
    if(num_hits > 100 ){
      goto punt;
    }
	  p7_bg_SetLength(bg, (int) chu->L[i]);            // TODO: remove need for cast
	  p7_oprofile_ReconfigLength(om, (int) chu->L[i]); //         (ditto)
	  
	  //	  printf("seq %d %s\n", chu->i0+i, chu->name[i]);
    float ssv_score;
printf("Sequence %s, ", chu->name[i]);
    p7_SSVFilter_shell_sse(chu->dsq[i], chu->L[i], om, eng->fx ,&ssv_score, card_OPROFILE, card_FILTERMX, num_hits);


	  p7_engine_Reuse(eng);
 /*   if (num_hits %100000 == 0){
      printf("processed %ld sequences\n", num_hits);
    } */
    num_hits++;
	}
 punt:
      esl_dsqdata_Recycle(dd, chu);
    }
    printf("Saw %ld sequences\n", num_hits);
  hipProfilerStop();
  esl_dsqdata_Close(dd);
  p7_oprofile_Destroy(om);
  p7_profile_Destroy(gm);
  p7_hmm_Destroy(hmm);
  p7_bg_Destroy(bg);
  p7_hmmfile_Close(hfp);
  //esl_alphabet_Destroy(abc);
  esl_getopts_Destroy(go);
  //destroy_oprofile_on_card(om, card_OPROFILE);
  exit(0);
}





