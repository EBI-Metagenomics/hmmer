#include "hip/hip_runtime.h"
/* Functions to determine whether a system's hardware contains one
 * or more NVIDIA GPUs that can run HMMER's CUDA acceleration
 * 
 * Contents:
 *   1. The p7_cuda object
 *   2. Functions to query CUDA cards
 *   5. Unit tests
 *   6. Test driver
 *   7. Example
 */

#include "easel.h"
#include "hmmer.h"
#include "p7_config.h"
#include "p7_cuda_error.h"

int p7_configure_cuda(P7_CUDA_CONFIG *ret_config){
#ifndef eslENABLE_CUDA // if we weren't compiled with CUDA support, never find any cards 
	ret_config->num_cards =0;
	ret_config->card_sms = NULL;
	return eslOK;
#endif
	int num_cards=0;
	size_t free_mem;
	size_t mask = 0xffffffffe0000000; // AND-ing with this should round down to half-gigabyte boundary
    p7_cuda_wrapper(hipGetDeviceCount(&num_cards));
    num_cards = 1; //debugging hack
printf("%d CUDA cards detected\n", num_cards);
	ret_config->num_cards = num_cards;

	if(num_cards == 0){
		ret_config->card_sms = 0;
		return eslOK;
	}
	else{
		ret_config->card_sms = (uint32_t *) malloc(num_cards * sizeof(uint32_t));
		ret_config->card_mem_sizes = (size_t *) malloc(num_cards * sizeof(size_t));
		ret_config->shared_per_block = (size_t *)malloc(num_cards * sizeof(size_t));
		ret_config->reg_per_block =(uint32_t *)malloc(num_cards * sizeof(uint32_t));
		ret_config->reg_per_sm =(uint32_t *)malloc(num_cards * sizeof(uint32_t));
		ret_config->threads_per_warp =(uint32_t *)malloc(num_cards * sizeof(uint32_t));
        ret_config->warps_per_block =(uint32_t *)malloc(num_cards * sizeof(uint32_t));
        ret_config->card_mem = (P7_CUDA_CARDMEM *)malloc(num_cards * sizeof(P7_CUDA_CARDMEM *));
        if ((ret_config->card_sms == NULL) ||
        	(ret_config->card_mem_sizes == NULL) ||
            (ret_config->shared_per_block == NULL) ||
            (ret_config->reg_per_block == NULL) ||
            (ret_config->reg_per_sm == NULL) ||
			(ret_config->threads_per_warp == NULL) ||
            (ret_config->warps_per_block == NULL) ||
            (ret_config->card_mem==NULL)) {
                goto ERROR;
        	}
        hipDeviceProp_t card_properties;
		for(int i = 0; i < num_cards; i++){
            printf("Handling card %d\n",i);
			//Get the information we need about the card
			hipGetDeviceProperties(&card_properties, i);
			ret_config->card_sms[i] = card_properties.multiProcessorCount;
			ret_config->shared_per_block[i] = card_properties.sharedMemPerBlock;
			ret_config->reg_per_block[i] = card_properties.regsPerBlock;
			ret_config->reg_per_sm[i] = card_properties.regsPerMultiprocessor;
			ret_config->threads_per_warp[i] = card_properties.warpSize;
			if(card_properties.warpSize != 32){
				printf("Your CUDA card %d reports that it has %d threads per warp.  HMMER currently only runs on CUDA cards with 32 threads/warp, so CUDA acceleration is being disabled.\n", i, card_properties.warpSize);
				goto ERROR;
			}
			if(ret_config->reg_per_block[i] < ret_config->reg_per_sm[i]){
				ret_config->warps_per_block[i] = ret_config->reg_per_block[i]/(P7_CUDA_REG_PER_THREAD *ret_config->threads_per_warp[i]);
			}
			else{
				ret_config->warps_per_block[i] = ret_config->reg_per_block[i] /(P7_CUDA_REG_PER_THREAD *
                ret_config->threads_per_warp[i]);
            }
			//hipMemGetInfo(&free_mem, &total);
        	ret_config->card_mem_sizes[i] = free_mem & mask;

            // allocate the buffers we'll use to transfer data to/from the
            // card. use regular malloc for the arrays that hold the pointers
            // to the buffers because we won't be copying them to the card
            // frequently
            ret_config->card_mem[i].num_streams = NUM_STREAMS;
            ret_config->card_mem[i].cpu_offsets = (uint64_t **)malloc(ret_config->card_mem[i].num_streams * sizeof(uint64_t *));
            ret_config->card_mem[i].gpu_offsets = (uint64_t **)malloc(ret_config->card_mem[i].num_streams * sizeof(uint64_t *));
            ret_config->card_mem[i].cpu_data = (char **)malloc(ret_config->card_mem[i].num_streams * sizeof(char *));
            ret_config->card_mem[i].cpu_data2 = (char **)malloc(ret_config->card_mem[i].num_streams * sizeof(char *));
            ret_config->card_mem[i].num_sequences = (uint32_t *)malloc(ret_config->card_mem[i].num_streams * sizeof(uint32_t));
            ret_config->card_mem[i].gpu_data =(char **)malloc(ret_config->card_mem[i].num_streams * sizeof(char *));
            ret_config->card_mem[i].cpu_hits = (int8_t **)malloc(ret_config->card_mem[i].num_streams * sizeof(int8_t *));
            ret_config->card_mem[i].gpu_hits = (int8_t **)malloc(ret_config->card_mem[i].num_streams * sizeof(int8_t *));
            ret_config->card_mem[i].cpu_scores = (float **)malloc(ret_config->card_mem[i].num_streams * sizeof(float *));
            ret_config->card_mem[i].gpu_scores = (float **)malloc(ret_config->card_mem[i].num_streams * sizeof(float *));
            ret_config->card_mem[i].cpu_lengths = (uint64_t **)malloc(ret_config->card_mem[i].num_streams * sizeof(uint64_t *));
            ret_config->card_mem[i].gpu_lengths = (uint64_t **)malloc(ret_config->card_mem[i].num_streams * sizeof(uint64_t *));
            ret_config->card_mem[i].cpu_sequences = (char ***)malloc(ret_config->card_mem[i].num_streams * sizeof(char **));
            ret_config->card_mem[i].streams = (hipStream_t *)malloc(ret_config->card_mem[i].num_streams * sizeof(hipStream_t));

            if ((ret_config->card_mem[i].cpu_offsets == NULL) ||
                (ret_config->card_mem[i].gpu_offsets == NULL) ||
                (ret_config->card_mem[i].cpu_scores == NULL) ||
                (ret_config->card_mem[i].gpu_scores == NULL) ||
                (ret_config->card_mem[i].num_sequences == NULL) ||
                (ret_config->card_mem[i].cpu_data == NULL) ||
                (ret_config->card_mem[i].gpu_data == NULL) ||
                (ret_config->card_mem[i].cpu_hits == NULL) ||
                (ret_config->card_mem[i].gpu_hits == NULL) ||
                (ret_config->card_mem[i].cpu_sequences == NULL) ||
                (ret_config->card_mem[i].streams == NULL)) {
                p7_Fail((char *)"Unable to allocate memory in p7_configure_cudan");
                }

            // cpu_num_hits and gpu_num_hits get allocated using
            // hipHostMalloc because they're one-d arrays
            p7_cuda_wrapper(hipHostMalloc((void **)&(ret_config->card_mem[i].cpu_num_hits),(ret_config->card_mem[i].num_streams * sizeof(uint32_t))));

            p7_cuda_wrapper(hipHostMalloc((void **)&(ret_config->card_mem[i].gpu_num_hits),(ret_config->card_mem[i].num_streams * sizeof(uint32_t))));

            // allocate the actual buffers. Use hipHostMalloc here for
            // buffers on the CPU side to pin the buffers in RAM, which
            // improves copy performance
            printf("allocating buffers for streams\n");
            for (int j = 0; j < ret_config->card_mem[i].num_streams; j++) {
printf("Handling stream %d\n", j);
                p7_cuda_wrapper(hipHostMalloc((void **)&(ret_config->card_mem[i].cpu_offsets[j]), (MAX_SEQUENCES * sizeof(uint64_t))));
printf("1\n");
                p7_cuda_wrapper(hipMalloc((void **)&(ret_config->card_mem[i].gpu_offsets[j]), (MAX_SEQUENCES * sizeof(uint64_t))));
printf("2\n");
                p7_cuda_wrapper(hipHostMalloc((void **)&(ret_config->card_mem[i].cpu_data[j]), DATA_BUFFER_SIZE));
printf("3\n");
                p7_cuda_wrapper(hipHostMalloc((void **)&(ret_config->card_mem[i].cpu_data2[j]), DATA_BUFFER_SIZE));
printf("4\n");
                p7_cuda_wrapper(hipMalloc((void **)&(ret_config->card_mem[i].gpu_data[j]), DATA_BUFFER_SIZE));
printf("a\n");
                p7_cuda_wrapper(hipHostMalloc((void **)&(ret_config->card_mem[i].cpu_hits[j]), (MAX_SEQUENCES * sizeof(int8_t))));

                p7_cuda_wrapper(hipMalloc((void **)&(ret_config->card_mem[i].gpu_hits[j]), (MAX_SEQUENCES * sizeof(int8_t))));
                p7_cuda_wrapper(hipHostMalloc((void **)&(ret_config->card_mem[i].cpu_scores[j]), (MAX_SEQUENCES * sizeof(float))));

                p7_cuda_wrapper(hipMalloc((void **)&(ret_config->card_mem[i].gpu_scores[j]), (MAX_SEQUENCES * sizeof(float))));
printf("b\n");
                p7_cuda_wrapper(hipHostMalloc((void **)&(ret_config->card_mem[i].cpu_sequences[j]), (MAX_SEQUENCES * sizeof(char *))));

                p7_cuda_wrapper(hipHostMalloc((void **)&(ret_config->card_mem[i].cpu_lengths[j]), (MAX_SEQUENCES * sizeof(uint64_t))));

                p7_cuda_wrapper(hipMalloc((void **)&(ret_config->card_mem[i].gpu_lengths[j]), (MAX_SEQUENCES * sizeof(uint64_t))));
printf("c\n");
                p7_cuda_wrapper(hipStreamCreate(&(ret_config->card_mem[i].streams[j])));
            }
            printf("Done with buffers\n");
        }
    }
    printf("leaving p7_configure_cuda\n");
	return eslOK;

ERROR:
//FIXME: free memory on error
  ret_config->num_cards = 0;
  ret_config->card_sms = NULL;
  ret_config->shared_per_block = NULL;
  ret_config->card_mem_sizes = NULL;
  ret_config->reg_per_block = NULL;
  ret_config->reg_per_sm = NULL;
  ret_config->shared_per_block = NULL;
  ret_config->threads_per_warp = NULL;
  ret_config->warps_per_block = NULL;
  return eslEMEM;
}

void p7_cuda_config_Destroy(P7_CUDA_CONFIG *cuda_config){
	if(cuda_config == NULL){
		return;
	}
	if(cuda_config->card_sms != NULL){
		free(cuda_config->card_sms);
	}
	free(cuda_config);
}

// Treats the elements of value across the threads in a warp as a vector of packed 16-bit integers
// Shifts that vector right one position and shifts in shiftin at the low end
// "Right" here is defined in HMMER notation as corresponding to an order where the low element of the 
// vector is written on the left of a string, so the resulting value at each thread is the low 16
// bits of that thread's value shifted up and ORed with the high 16 bits of the value from the next lower-numbered
// thread.  Thread 0 has shiftin is those low bits.
__device__ inline unsigned int esl_cuda_rightshift_int16(unsigned int value, int16_t shiftin){
    unsigned int temp = __shfl_up_sync(0xffffffff, value, 1);
    temp = __byte_perm(temp, value, 0x1076);
    if(threadIdx.x == 0){
        temp = __byte_perm(temp, shiftin, 3254);
    }
    return temp;
}

//returns the largest element of a vector of packed 16-bit signed integers distributed across a warp 
__device__ inline int16_t esl_cuda_hmax_epi16(unsigned int vector){
    // First, get the element-wise max of the value on each pair of threads
    unsigned int temp1 =__vmaxs2(__shfl_sync(0xffffffff, vector, 0, 2), __shfl_sync(0xffffffff, vector, 1, 2));
    // Then, each quad.  Use a second variable to prevent race conditions
    unsigned int temp2 = __vmaxs2(__shfl_sync(0xffffffff, temp1, 0, 4), __shfl_sync(0xffffffff, temp1, 2, 4));
    // Next, each 8-thread group
    temp1 = __vmaxs2(__shfl_sync(0xffffffff, temp2, 0, 8), __shfl_sync(0xffffffff, temp2, 4, 8));
    // 16-thread group
    temp2 = __vmaxs2(__shfl_sync(0xffffffff, temp1, 0, 16), __shfl_sync(0xffffffff, temp1, 8, 16));
    // Full warp
    temp1 = __vmaxs2(__shfl_sync(0xffffffff, temp2, 0, 32), __shfl_sync(0xffffffff, temp2, 16, 32));
    
    temp2 = __vmaxs2(temp1, temp1 >> 16); // low 16 bits now has the larger of the two elements
    return((int16_t) (temp2 & 0xffff));
}