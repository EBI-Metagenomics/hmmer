#include "hip/hip_runtime.h"
#include "hmmer.h"
#include "esl_red_black.h"
#include "easel.h"
#include "ssv_cuda.h"
#include "p7_cuda.h"
#include "p7_orion.h"
#include "p7_cuda_error.h"
//#define __NVVP_PROFILE_ON  //add events for better profiling 

#ifdef __NVVP_PROFILE_ON
#include "nvToolsExt.h"
#include "nvToolsExtCudaRt.h"
#endif

#define BACKEND_SWITCH_THRESHOLD 1000000



char * restripe_char(char *source, int source_chars_per_vector, int dest_chars_per_vector, int source_length, int *dest_length);

__global__
void SSV_cuda(int num_sequences, const __restrict__ uint8_t *data, const __restrict__ uint64_t *lengths, const __restrict__ uint64_t *offsets, __restrict__ uint64_t *hits, P7_OPROFILE *om, double mu, double lambda);


static void p7_cuda_worker_thread_back_end_sequence_search_loop(P7_DAEMON_WORKERNODE_STATE *workernode,  uint32_t my_id, int cleanup);
void send_sequence_chunk_to_cuda_card(P7_DAEMON_WORKERNODE_STATE *workernode, P7_CUDA_CONFIG *cuda_config, uint32_t my_id, uint64_t *chunk_end, uint64_t work_end, char **sequence_data, int stream, dim3 threads_per_block, dim3 num_blocks, P7_OPROFILE *om, double mu, double lambda);

void parse_CUDA_chunk_results(P7_DAEMON_WORKERNODE_STATE *workernode, P7_CUDA_CONFIG *cuda_config, uint32_t my_id, uint32_t stream);
// GPU kernel that copies values from the CPU version of an OPROFILE to one on the GPU.  Should generally only be called on one GPU core
__global__ void copy_oprofile_values_to_card(P7_OPROFILE *the_profile, float tauBM, float scale_b, float scale_w, int16_t base_w, int16_t ddbound_w, int L, int M, int V, int max_length, int allocM, int allocQb, int allocQw, int allocQf, int mode, float nj, int is_shadow, int8_t **rbv){

  the_profile->tauBM = tauBM;
  the_profile->scale_b = scale_b;
  the_profile->scale_w = scale_w;
  the_profile->base_w = base_w;
  the_profile->ddbound_w = ddbound_w;
  the_profile->L = L;
  the_profile->M = M;
  the_profile->V = V;
  the_profile->max_length = max_length;
  the_profile->allocM = allocM;
  the_profile->allocQb = allocQb;
  the_profile->allocQw = allocQw;
  the_profile->allocQf = allocQf;
  the_profile->mode = mode;
  the_profile->nj = nj;
  the_profile->is_shadow = is_shadow;
  the_profile->rbv = rbv;
}


// GPU kernel that initializes a filtermx structure
__global__ void initialize_filtermx_on_card(P7_FILTERMX *the_filtermx){
  the_filtermx->M = 0;
  the_filtermx->Vw = 64; // 32 cores * 32 bits = 1024 bits = 128 bytes = 64 * 16 bits
  the_filtermx->allocM = 0;
  the_filtermx->dp = NULL;
  the_filtermx->type = p7F_SSVFILTER;
}


// allocates and populates a P7_OPROFILE structure on a CUDA card that matches the one passed as its argument
P7_OPROFILE *create_oprofile_on_card(P7_OPROFILE *the_profile){
  P7_OPROFILE *cuda_OPROFILE;

  int Q = P7_Q(the_profile->M, the_profile->V);

  p7_cuda_wrapper(hipMalloc(&cuda_OPROFILE, sizeof(P7_OPROFILE)));

  // allocate and copy over rbv 2-D array
  unsigned int **cuda_rbv;
  p7_cuda_wrapper(hipMalloc(&cuda_rbv, the_profile->abc->Kp * sizeof(unsigned int *)));
  int i;
  char *restriped_rbv;
  int restriped_rbv_size;

  unsigned int **cuda_rbv_temp = cuda_rbv; // use this variable to copy rbv pointers into CUDA array 
  for(i = 0; i < the_profile->abc->Kp; i++){
    int *cuda_rbv_entry;
  restriped_rbv = restripe_char ((char*)(the_profile->rbv[i]), the_profile->V, 128, Q * the_profile->V, &restriped_rbv_size);
  //restriped_rbv = (int *) restripe_char((char *)(the_profile->rbv[i]), the_profile->V, 128, Q * the_profile->V, &restriped_rbv_size);

    p7_cuda_wrapper(hipMalloc(&cuda_rbv_entry, restriped_rbv_size));

    p7_cuda_wrapper(hipMemcpy(cuda_rbv_entry, restriped_rbv, restriped_rbv_size, hipMemcpyHostToDevice));

    p7_cuda_wrapper(hipMemcpy(cuda_rbv_temp, &cuda_rbv_entry, sizeof(int *) , hipMemcpyHostToDevice));
    cuda_rbv_temp +=1;
  }
 

  // copy over base parameters.  Only call this kernel on one core because it just assigns values to fields in the data structure and has no parallelism
  copy_oprofile_values_to_card<<<1,1>>>(cuda_OPROFILE, the_profile->tauBM, the_profile->scale_b, the_profile->scale_w, the_profile->base_w, the_profile->ddbound_w, the_profile->L, the_profile->M, the_profile->V, the_profile->max_length, the_profile->allocM, the_profile->allocQb, the_profile->allocQw, the_profile->allocQf, the_profile->mode, the_profile->nj, the_profile->is_shadow, (int8_t **) cuda_rbv);
  p7_kernel_error_check();
 return cuda_OPROFILE;
}

void destroy_oprofile_on_card(P7_OPROFILE *cpu_oprofile, P7_OPROFILE *cuda_oprofile){
  int i;
  for(i = 0; i < cpu_oprofile->abc->Kp; i++){
    p7_cuda_wrapper(hipFree(cuda_oprofile->rbv[i]));
  }
  p7_cuda_wrapper(hipFree(cuda_oprofile->rbv));
  p7_cuda_wrapper(hipFree(cuda_oprofile));
}

P7_FILTERMX *create_filtermx_on_card(){
  P7_FILTERMX *the_filtermx;
  
  p7_cuda_wrapper(hipMalloc(&the_filtermx, sizeof(P7_FILTERMX)));
  initialize_filtermx_on_card<<<1,1>>>(the_filtermx);
  p7_kernel_error_check();
  return the_filtermx;
}


char * restripe_char(char *source, int source_chars_per_vector, int dest_chars_per_vector, int source_length, int *dest_length){
  char *dest;
  int dest_num_vectors, unpadded_dest_vectors;
  int source_num_vectors;
  source_num_vectors = source_length/source_chars_per_vector;
  if(source_num_vectors * source_chars_per_vector != source_length){
    source_num_vectors++;
  }
  unpadded_dest_vectors = source_length/dest_chars_per_vector;
  if(unpadded_dest_vectors * dest_chars_per_vector != source_length){
    unpadded_dest_vectors++;  //round up if source length isn't a multiple of the dest vector length
  }
 // printf("Unpadded_dest_vectors = %d. ", unpadded_dest_vectors);
  dest_num_vectors = unpadded_dest_vectors + MAX_BAND_WIDTH -1; // add extra vectors for SSV wrap-around

  dest = (char *) malloc(dest_num_vectors * dest_chars_per_vector);
  *dest_length = dest_num_vectors * dest_chars_per_vector;
  int source_pos, dest_pos;
  int i;

  for(i = 0; i < source_length; i++){
    source_pos = ((i % source_num_vectors) * source_chars_per_vector) + (i / source_num_vectors);
    dest_pos = ((i % unpadded_dest_vectors) * dest_chars_per_vector) + (i / unpadded_dest_vectors);
    dest[dest_pos] = (int) source[source_pos];
  }

  // pad out the dest vector with zeroes if necessary
  for(; i < unpadded_dest_vectors * dest_chars_per_vector; i++){
      dest_pos = ((i % unpadded_dest_vectors) * dest_chars_per_vector) + (i / unpadded_dest_vectors);
    //  printf("Padding 0 at location %d \n", dest_pos);
    dest[dest_pos] = -128;
  }

  // add the extra copies of the early vectors to support the SSV wrap-around
  for(int source = 0; i < dest_num_vectors * dest_chars_per_vector; i++){
    dest[i] = dest[source];
   // printf("Padding from location %d to location %d\n", source, i);
    source++;
  }

  return dest;

}
int16_t *restripe_vitfilter(int16_t *source, int source_int16s_per_vector, int dest_int16s_per_vector, int source_length, int *dest_length){
  int dest_num_vectors;
  int source_num_vectors;
  int16_t *dest;
  source_num_vectors = (source_length /sizeof(int16_t)) / source_int16s_per_vector;
  if (source_num_vectors * source_int16s_per_vector *sizeof(int16_t)!= source_length) {
    source_num_vectors++;
  }
  dest_num_vectors = (source_length/sizeof(int16_t)) / dest_int16s_per_vector;
  if (dest_num_vectors * dest_int16s_per_vector * sizeof(int16_t) != source_length) {
    dest_num_vectors++; // round up if source length isn't a multiple of
                             // the dest vector length
  }

  *dest_length = dest_num_vectors * dest_int16s_per_vector * sizeof(int16_t);
  dest = (int16_t *)malloc(*dest_length);
  int source_pos, dest_pos;
  int i;

  for (i = 0; i < (source_length / sizeof(int16_t)); i++) {
    source_pos = ((i % source_num_vectors) * source_int16s_per_vector) +
                 (i / source_num_vectors);
    dest_pos = ((i % dest_num_vectors) * dest_int16s_per_vector) +
               (i / dest_num_vectors);
    dest[dest_pos] = source[source_pos];
  }

  // pad out the dest vector with zeroes if necessary
  for (; i < dest_num_vectors * dest_int16s_per_vector; i++) {
    dest_pos = ((i % dest_num_vectors) * dest_int16s_per_vector) +
               (i / dest_num_vectors);
    //  printf("Padding 0 at location %d \n", dest_pos);
    dest[dest_pos] = -32768;
  }

  return(dest);
}

float *restripe_fwdfilter(float *source, int source_floats_per_vector,
                            int dest_floats_per_vector, int source_length,
                            int *dest_length) {
  int dest_num_vectors;
  int source_num_vectors;
  float *dest;
  source_num_vectors =
      (source_length / sizeof(float)) / source_floats_per_vector;
  if (source_num_vectors * source_floats_per_vector *sizeof(float) != source_length) {
    source_num_vectors++;
  }
  dest_num_vectors = (source_length / sizeof(float)) / dest_floats_per_vector;
  if (dest_num_vectors * dest_floats_per_vector * sizeof(float) != source_length) {
    dest_num_vectors++; // round up if source length isn't a multiple of
                        // the dest vector length
  }

  *dest_length = dest_num_vectors * dest_floats_per_vector * sizeof(float);
  dest = (float *)malloc(*dest_length);
  int source_pos, dest_pos;
  int i;

  for (i = 0; i < (source_length / sizeof(float)); i++) {
    source_pos = ((i % source_num_vectors) * source_floats_per_vector) +
                 (i / source_num_vectors);
    dest_pos = ((i % dest_num_vectors) * dest_floats_per_vector) +
               (i / dest_num_vectors);
    dest[dest_pos] = source[source_pos];
  }

  // pad out the dest vector with zeroes if necessary
  for (; i < dest_num_vectors * dest_floats_per_vector; i++) {
    dest_pos = ((i % dest_num_vectors) * dest_floats_per_vector) +
               (i / dest_num_vectors);
    //  printf("Padding 0 at location %d \n", dest_pos);
    dest[dest_pos] = -32768;
  }

  return (dest);
}

int p7_cuda_worker_thread_front_end_sequence_search_loop(P7_DAEMON_WORKERNODE_STATE *workernode, uint32_t my_id,
P7_CUDA_CONFIG *cuda_config, P7_OPROFILE *om, double mu, double lambda);


extern "C" P7_BACKEND_QUEUE_ENTRY *workernode_get_backend_queue_entry_from_pool(P7_DAEMON_WORKERNODE_STATE *workernode);
extern "C" ESL_RED_BLACK_DOUBLEKEY *workernode_get_hit_list_entry_from_pool(P7_DAEMON_WORKERNODE_STATE *workernode, uint32_t my_id);
extern "C" P7_BACKEND_QUEUE_ENTRY *workernode_get_backend_queue_entry_from_queue(P7_DAEMON_WORKERNODE_STATE *workernode);
extern "C" void workernode_increase_backend_threads(P7_DAEMON_WORKERNODE_STATE *workernode);
extern "C" void workernode_put_backend_queue_entry_in_pool(P7_DAEMON_WORKERNODE_STATE *workernode, P7_BACKEND_QUEUE_ENTRY *the_entry);
extern "C" uint64_t worker_thread_get_chunk(P7_DAEMON_WORKERNODE_STATE *workernode, uint32_t my_id, volatile uint64_t *start, volatile uint64_t *end);


extern "C" void workernode_put_backend_queue_entry_in_queue(P7_DAEMON_WORKERNODE_STATE *workernode, P7_BACKEND_QUEUE_ENTRY *the_entry);
extern "C" void workernode_put_backend_chain_in_queue(P7_DAEMON_WORKERNODE_STATE *workernode, int chain_length, P7_BACKEND_QUEUE_ENTRY *chain_start, P7_BACKEND_QUEUE_ENTRY *chain_end);
void *p7_server_cuda_worker_thread(void *worker_argument){
  int stop;


  // unpack the box that is the pthread single argument
  P7_DAEMON_WORKER_ARGUMENT *my_argument = (P7_DAEMON_WORKER_ARGUMENT *) worker_argument;
  uint32_t my_id = my_argument->my_id;
  P7_DAEMON_WORKERNODE_STATE *workernode = my_argument->workernode;
  P7_CUDA_CONFIG *cuda_config = workernode->cuda_config;
  hipSetDevice(my_id);
  printf("thread %d starting as cuda thread\n", my_id);

  // create the engine object we'll use 
  ESL_ALPHABET *temp_abc = esl_alphabet_Create(eslAMINO); // All we use the alphabet for in engine_Create is setting the size of the
  // wrkKp field, so use the biggest alphabet 

  if(temp_abc == NULL){
    p7_Fail((char *) "Unable to allocate memory in p7_server_worker_thread\n");
  }
  P7_ENGINE_STATS *engine_stats = p7_engine_stats_Create();
  if(engine_stats == NULL){
    p7_Fail((char *) "Unable to allocate memory in p7_server_worker_thread\n");
  }
  workernode->thread_state[my_id].engine = p7_engine_Create(temp_abc, NULL, engine_stats, 400, 400);
  if(workernode->thread_state[my_id].engine == NULL){
    p7_Fail((char *) "Unable to allocate memory in p7_server_worker_thread\n");
  }

  // Allocate a pool of empty hit objects
  workernode->thread_state[my_id].empty_hit_pool = p7_hitlist_entry_pool_Create(HITLIST_POOL_SIZE);
   if(workernode->thread_state[my_id].empty_hit_pool == NULL){
    p7_Fail((char *) "Unable to allocate memory in p7_server_worker_thread\n");
  }

  // Tell the master thread that we're awake and ready to go
  if(pthread_mutex_lock(&(workernode->wait_lock))){  // Use blocking lock here because we may be waiting a while
    p7_Fail((char *) "Couldn't acquire wait_lock mutex in p7_server_worker_thread");
  }

  workernode->num_waiting +=1;  //mark that we're now waiting for the go signal

  pthread_cond_wait(&(workernode->start), &(workernode->wait_lock)); // wait until master tells us to go

  pthread_mutex_unlock(&(workernode->wait_lock));  // We come out of pthread_cond_wait holding the lock,
  // need to release it to let the next thread go
  
  // Main work loop.  The thread remains in this loop until it is told to terminate.
  while(!workernode->shutdown){
    switch(workernode->search_type){ // do the right thing for each search type
      case SEQUENCE_SEARCH:
      case SEQUENCE_SEARCH_CONTINUE:

        // Create any models we need. Check every time to avoid race condition between requests for more work at the start of a search
        // and threads starting up. 
        if(workernode->thread_state[my_id].bg == NULL){
          workernode->thread_state[my_id].bg = p7_bg_Create(workernode->compare_model->abc);
          if(workernode->thread_state[my_id].bg == NULL){
            p7_Fail((char *) "Unable to allocate memory in p7_server_worker_thread\n");
          }
        }
        if(workernode->thread_state[my_id].gm == NULL){
          workernode->thread_state[my_id].gm = p7_profile_Create (workernode->compare_model->M, workernode->compare_model->abc);
          if(workernode->thread_state[my_id].gm == NULL){
            p7_Fail((char *) "Unable to allocate memory in p7_server_worker_thread\n");
          }
          p7_profile_Copy(workernode->compare_model, workernode->thread_state[my_id].gm);
        }
        if(workernode->thread_state[my_id].om == NULL){
          workernode->thread_state[my_id].om = p7_oprofile_Create(workernode->thread_state[my_id].gm->M, workernode->thread_state[my_id].gm->abc);      
          if(workernode->thread_state[my_id].om == NULL){
            p7_Fail((char *) "Unable to allocate memory in p7_server_worker_thread\n");
          }

          p7_oprofile_Convert (workernode->thread_state[my_id].gm, workernode->thread_state[my_id].om);

          p7_bg_SetFilter(workernode->thread_state[my_id].bg, workernode->thread_state[my_id].om->M, workernode->thread_state[my_id].om->compo);
        }

        P7_OPROFILE *card_OPROFILE;
        card_OPROFILE = create_oprofile_on_card(workernode->thread_state[my_id].om);
        //P7_FILTERMX *card_FILTERMX;
        //card_FILTERMX = create_filtermx_on_card();

        stop = 0;
        while(stop == 0){
          stop = p7_cuda_worker_thread_front_end_sequence_search_loop(workernode, my_id, cuda_config, card_OPROFILE, workernode->thread_state[my_id].om->evparam[p7_SMU], workernode->thread_state[my_id].om->evparam[p7_SLAMBDA]);
        }
        break;

      case HMM_SEARCH:
        p7_Fail((char *) "Hmmscan functionality disabled in this version\n");

         break;
      case IDLE:
         p7_Fail((char *) "Workernode told to start search of type IDLE");
         break;
    }
   while(pthread_mutex_trylock(&(workernode->wait_lock))){
      // spin-wait until the lock on the hitlist is cleared.  Should never be locked for long
    }

    workernode->num_waiting +=1;  //mark that we're now waiting for the go signal
    pthread_cond_wait(&(workernode->start), &(workernode->wait_lock)); // wait until master tells us to go

    pthread_mutex_unlock(&(workernode->wait_lock));  // We come out of pthread_cond_wait holding the lock
  }
  /*
  for(int i = 0; i < buffer_state.num_streams; i++){
    p7_cuda_wrapper(hipFree(buffer_state.gpu_offsets[i]));
    p7_cuda_wrapper(hipHostFree(buffer_state.cpu_offsets[i]));
    p7_cuda_wrapper(hipFree(buffer_state.gpu_data[i]));
    p7_cuda_wrapper(hipHostFree(buffer_state.cpu_data[i]));
    p7_cuda_wrapper(hipFree(buffer_state.gpu_hits[i]));
    p7_cuda_wrapper(hipHostFree(buffer_state.cpu_hits[i]));
    p7_cuda_wrapper(hipFree(buffer_state.gpu_lengths[i]));
    p7_cuda_wrapper(hipHostFree(buffer_state.cpu_lengths[i]));
    p7_cuda_wrapper(hipHostFree(buffer_state.cpu_sequences[i]));
  }
  free(buffer_state.gpu_offsets);
  free(buffer_state.cpu_offsets);
  free(buffer_state.cpu_data);
  free(buffer_state.gpu_data);
  free(buffer_state.cpu_hits);
  free(buffer_state.gpu_hits);
  free(buffer_state.gpu_lengths);
  free(buffer_state.cpu_lengths);
  free(buffer_state.cpu_sequences);
  free(buffer_state.streams);
  */
   // We've been shut down, so exit the thread
   printf("Exiting CUDA thread\n");
   pthread_exit(NULL);

}


#define ALIGNEIGHT_MASK 0xffffffffffffffe0 // mask off low three bits

int p7_cuda_worker_thread_front_end_sequence_search_loop(P7_DAEMON_WORKERNODE_STATE *workernode, uint32_t my_id, P7_CUDA_CONFIG *cuda_config, P7_OPROFILE *om, double mu, double lambda){
  uint64_t start,end;
  uint64_t seq_id=0;
  char *the_sequence, *data_start;
  int my_stream = 0;
  while(pthread_mutex_trylock(&(workernode->work[my_id].lock))){
    // spin-wait until the lock on our queue is cleared.  Should never be locked for long
    // Lock our work queue because get_chunk will update our start and end pointers
   }

  workernode->work[my_id].start = 0xffffffffffffffff; // We don't currently support stealing out of GPU thread queues, so tell everyone else 
  // our work queue is empty

  pthread_mutex_unlock(&(workernode->work[my_id].lock)); // release lock
  int warps_per_block;
  dim3 threads_per_block, num_blocks;
  num_blocks.x = workernode->cuda_config->card_sms[my_id]; // This counts on our convention that the GPU threads are the low-id threads
  //printf("Worker thread %d using %d SMs\n", my_id, num_blocks.x);
  num_blocks.y = 1;
  num_blocks.z = 1;
  warps_per_block = 32;
  threads_per_block.x = 32;
  threads_per_block.y = warps_per_block;
  threads_per_block.z = 1;
  int num_chunks_submitted = 0; 
  int num_chunks_parsed = 0;
  while(1){ // Iterate forever, we'll return from the function rather than exiting this loop

    // try to get some work from the global queue
    uint64_t work_on_global = worker_thread_get_chunk(workernode, my_id, &(start), &(end));
    // grab the start and end pointers from our work queue

    if(work_on_global){ //there was work left to gut
      seq_id = start; // set this to the start of the chunk to prevent problems when chunks are out of sequence order
  //    printf("GPU thread got work chunk of size %lu, start = %lu, end = %lu\n", end-start, start, end);
      // process the chunk of comparisons we got

      // get pointer to first sequence to search
      p7_shard_Find_Contents_Nexthigh(workernode->database_shards[workernode->compare_database], start,  &(data_start));

      the_sequence = data_start;
  
      // go through the sequences in our work chunk, creating the data buffers we'll send to the GPU
      // and sending them for processing
      while(seq_id <= end){
        // Step 1: figure out how many sequences will fit in a buffer and create
        // the vector of offsets to the start of each sequence in the buffer
      
       //empty out the backend queue if it gets too full before processing more front-end requests
        while(workernode->backend_queue_depth > BACKEND_SWITCH_THRESHOLD) {
          //printf("GPU thread switching to back-end processing\n");
          p7_cuda_worker_thread_back_end_sequence_search_loop(workernode, my_id, 0);
          //printf("GPU thread switching to front-end processing\n");
        }

        send_sequence_chunk_to_cuda_card(workernode, cuda_config, my_id, &seq_id, end, &the_sequence, my_stream, threads_per_block, num_blocks, om, mu, lambda);
        num_chunks_submitted +=1;

    //    printf("CUDA card processed chunk of %d sequences ending at %lu\n", num_sequences, seq_id);
/*        p7_cuda_wrapper(hipMemcpy(cuda_config->card_mem[my_id].cpu_data2[my_stream], cuda_config->card_mem[my_id].gpu_data[my_stream], current_offset, hipMemcpyDeviceToHost));
        if(memcmp(cuda_config->card_mem[my_id].cpu_data[my_stream], cuda_config->card_mem[my_id].cpu_data2[my_stream], current_offset) != 0){
          printf("sequence data appears to have been corrupted while in GPU\n");
        }  */  // uncomment this to check for data corruption on GPU side
        if(num_chunks_submitted >= cuda_config->card_mem[my_id].num_streams){
          //let the number of chunks we've submitted to the GPU get num_streams ahead before we start parsing
          parse_CUDA_chunk_results(workernode, cuda_config, my_id, (num_chunks_parsed % cuda_config->card_mem[my_id].num_streams));
          num_chunks_parsed++;
        }
        if (workernode->backend_queue_depth > (workernode->num_backend_threads << BACKEND_INCREMENT_FACTOR)){
          // There are too many back-end comparisons waiting in the queue, so switch a thread from frontend to backend
          workernode_increase_backend_threads(workernode);
        }
          
        
       //printf("GPU thread %d finished chunk with %d sequences and %d hits\n", my_id, num_sequences, num_hits);
        my_stream++;
        if(my_stream == cuda_config->card_mem[my_id].num_streams){
          my_stream = 0;
        }
      }

    }
    else{
      // Parse all the remaining unparsed chunks
      for(int i = num_chunks_parsed; i < num_chunks_submitted; i++){
        parse_CUDA_chunk_results(workernode, cuda_config, my_id, (i % cuda_config->card_mem[my_id].num_streams));
      }
      if(!work_on_global){
          if(workernode->backend_queue_depth != 0){
            // There are backend queue entries to process, so do one and then re-check
            p7_cuda_worker_thread_back_end_sequence_search_loop(workernode, my_id, 1);
            return 0;
        }
        return 1;
      }
    }
  }
}

// worker_thread_back_end_sequence_search_loop
/*! \brief Performs back-end computations when executing a one-sequence many-HMM search
 *  \details Iterates through the sequences in the back-end queue, performing the main stage of the engine on each sequence.
 *  Places any hits found in the thread's hit list.  Switches the thread to front-end mode and returns if there are no sequences
 *  remaining in the back-end queue.
 *  \param [in,out] workernode The node's P7_DAEMON_WORKERNODE_STATE object, which is modified during execution.
 *  \param [in] my_id The worker thread's id (index into arrays of thread-specific state).
 *  \returns nothing 
 *  \bug Currently treats any comparison that reaches the back end as a hit.  Needs to be updated with real hit detection.
 *  \bug Hits are always sorted by sequence ID.  Need to add an option to search by score when we have real score generation.
 */

// Note that this function uses a different return criteria than the CPU one and that the method for switching into and
// out of back-end mode is different because the GPU thread doesn't participate in the logic to choose the best thread 
// to switch into back-end mode that the CPU threads use.
static void p7_cuda_worker_thread_back_end_sequence_search_loop(P7_DAEMON_WORKERNODE_STATE *workernode, uint32_t my_id, int cleanup){
  //printf("GPU thread %d entering back-end mode with %d entries in queue\n", my_id, workernode->backend_queue_depth);
  // Grab a sequence to work on
  P7_BACKEND_QUEUE_ENTRY *the_entry = workernode_get_backend_queue_entry_from_queue(workernode);
  int entries_dequeued = 1;
  int entries_returned = 0;
  ESL_RED_BLACK_DOUBLEKEY *the_hit_entry;
  int overthruster_result = eslFAIL;
  while(the_entry != NULL){
  // There's a sequence in the queue, so do the backend comparison 

    // configure the model and engine for this comparison
    p7_bg_SetLength(workernode->thread_state[my_id].bg, the_entry->L);           
        p7_oprofile_ReconfigLength(workernode->thread_state[my_id].om, the_entry->L);
 
    if(the_entry->do_overthruster !=0){
      //need to do the overthruster part of this comparison, generally because CUDA doesn't do the full overthruster
        char *seqname;
        p7_shard_Find_Descriptor_Nexthigh(workernode->database_shards[workernode->compare_database], the_entry->seq_id, &seqname);
        overthruster_result = p7_engine_Overthruster_roundtwo(workernode->thread_state[my_id].engine, the_entry->sequence, the_entry->L, workernode->thread_state[my_id].om, workernode->thread_state[my_id].bg, the_entry->score, seqname, the_entry->seq_position, the_entry->seq_in_chunk, the_entry->seq_id);  
    }
    else{
      // don't do the overthruster, but do set up the sparse mask for the main stage
      // don't need to do this if we run the overthruster, as it will handle it
     P7_SPARSEMASK *temp_mask = the_entry->sm;
      the_entry->sm = workernode->thread_state[my_id].engine->sm;
      workernode->thread_state[my_id].engine->sm = temp_mask;
    }

    if((the_entry->do_overthruster == 0) || (overthruster_result != eslFAIL)){ // this comparison passed the overthruster, so do the main stage
      p7_profile_SetLength(workernode->thread_state[my_id].gm, the_entry->L);
      p7_engine_Main(workernode->thread_state[my_id].engine, the_entry->sequence, the_entry->L, workernode->thread_state[my_id].gm); 


#ifdef TEST_SEQUENCES 
        // Record that we processed this sequence
        workernode->sequences_processed[the_entry->seq_id] = 1;
#endif

    // Stub code that treats any comparison that reaches the back end as a hit

      the_hit_entry = workernode_get_hit_list_entry_from_pool(workernode, my_id);
      the_hit_entry->key = (double) the_entry->seq_id; // For now, we only sort on sequence ID.  Need to change this to possibly sort
      // on score

      // Fake up a hit for comparison purposes.  Do not use for actual analysis
      P7_HIT *the_hit = (P7_HIT *) the_hit_entry->contents;
      the_hit->seqidx = the_entry->seq_id;
      the_hit->sortkey = the_hit_entry->key; // need to fix this to sort on score when we make hits work
      char *descriptors;

      // Get the descriptors for this sequence
      p7_shard_Find_Descriptor_Nexthigh(workernode->database_shards[workernode->compare_database], the_entry->seq_id, &descriptors);
      the_hit->name = descriptors;
      the_hit->acc = descriptors + (strlen(the_hit->name) +1); //+1 for termination character
      the_hit->desc = the_hit->acc + (strlen(the_hit->acc) +1); //+1 for termination character

      // Add the hit to the threads's list of hits
      while(pthread_mutex_trylock(&(workernode->thread_state[my_id].hits_lock))){
        // spin-wait until the lock on the hitlist is cleared.  Should never be locked for long
      }                
      the_hit_entry->large = workernode->thread_state[my_id].my_hits;
      workernode->thread_state[my_id].my_hits = the_hit_entry;
      pthread_mutex_unlock(&(workernode->thread_state[my_id].hits_lock));
    }

    // Done with the comparison, reset for next time
    overthruster_result = eslFAIL;
    workernode_put_backend_queue_entry_in_pool(workernode, the_entry); // Put the entry back in the free pool
    entries_returned++;
    p7_engine_Reuse(workernode->thread_state[my_id].engine);  // Reset engine structure for next comparison

  
    if((workernode->backend_queue_depth < BACKEND_SWITCH_THRESHOLD / 2) && (cleanup == 0)){ // The backend queue has shrunk enough that we should go
      // back to processing filters in CUDA
     //printf("GPU thread %d leaving back-end mode with %d entries in backend queue\n", my_id, workernode->backend_queue_depth);
      if(entries_dequeued != entries_returned){
        printf("CUDA backend search loop dequeued %d entries but returned %d\n", entries_dequeued, entries_returned);
      }
      return;
    }
    the_entry = workernode_get_backend_queue_entry_from_queue(workernode); //see if there's another backend operation to do
    entries_dequeued++;
  }
  // Should only get here very rarely, in the odd case where the backend queue goes from over the threshold to empty very quickly
  return;
}


void send_sequence_chunk_to_cuda_card(P7_DAEMON_WORKERNODE_STATE *workernode, P7_CUDA_CONFIG *cuda_config, uint32_t my_id, uint64_t *chunk_end, uint64_t work_end, char **sequence_data, int stream, dim3 threads_per_block, dim3 num_blocks, P7_OPROFILE *om, double mu, double lambda){
#ifdef __NVVP_PROFILE_ON //define the events we'll use in profiling
  char prep_event_string[] = "Prep -1111111111";
  nvtxEventAttributes_t prep_event = {0};
  prep_event.version = NVTX_VERSION;
  prep_event.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  prep_event.colorType = NVTX_COLOR_ARGB;
  prep_event.color = 0xffff00; //yellow
  prep_event.messageType = NVTX_MESSAGE_TYPE_ASCII;
  prep_event.message.ascii = prep_event_string;

  char send_event_string[] = "Send -1111111111";
  nvtxEventAttributes_t send_event = {0};
  send_event.version = NVTX_VERSION;
  send_event.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  send_event.colorType = NVTX_COLOR_ARGB;
  send_event.color = 0xff8000; //orange
  send_event.messageType = NVTX_MESSAGE_TYPE_ASCII;
  send_event.message.ascii = send_event_string;

  char comp_event_string[] = "Comp -1111111111";
  nvtxEventAttributes_t comp_event = {0};
  comp_event.version = NVTX_VERSION;
  comp_event.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  comp_event.colorType = NVTX_COLOR_ARGB;
  comp_event.color = 0x0000FF; //blue
  comp_event.messageType = NVTX_MESSAGE_TYPE_ASCII;
  comp_event.message.ascii = comp_event_string;


  char recv_event_string[] = "Recv -1111111111";
  nvtxEventAttributes_t recv_event = {0};
  recv_event.version = NVTX_VERSION;
  recv_event.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  recv_event.colorType = NVTX_COLOR_ARGB;
  recv_event.color = 0x0000FF; //green
  recv_event.messageType = NVTX_MESSAGE_TYPE_ASCII;
  recv_event.message.ascii = recv_event_string;
#endif 


  uint64_t seq_id;
  uint32_t num_sequences = 0;
  uint64_t L, L_effective;
  char *the_sequence = *sequence_data;
  char *sequence_start = the_sequence;
  uint64_t current_offset = 0;
  seq_id = *((uint64_t *) the_sequence);
  the_sequence += sizeof(uint64_t);
  L = *((uint64_t *) the_sequence);
  the_sequence += sizeof(uint64_t);

  // Round sequence length up to next multiple of eight
  L_effective = (L + 31) & ALIGNEIGHT_MASK; 
#ifdef __NVVP_PROFILE_ON
  sprintf(prep_event_string, "Prep %d", stream);  // This will fail miserably if stream is more than 10 digits, so don't do that
  nvtxRangeId_t prep = nvtxRangeStartEx(&prep_event);
#endif
  while((seq_id <= work_end) &&(num_sequences < MAX_SEQUENCES) && ((current_offset + L_effective) < DATA_BUFFER_SIZE)){
    //There's room in the buffer for the next sequence.  Note that this assumes that the data buffer
    //is larger than the longest possible sequence, so DATA_BUFFER_SIZE should not be made less than 100K
    cuda_config->card_mem[my_id].cpu_lengths[stream][num_sequences] = L;
    cuda_config->card_mem[my_id].cpu_sequences[stream][num_sequences] = sequence_start;
    cuda_config->card_mem[my_id].cpu_offsets[stream][num_sequences] = current_offset;
    memcpy((cuda_config->card_mem[my_id].cpu_data[stream] + current_offset), the_sequence +1, L);
    /*      float retsc, nullsc;
          p7_bg_SetLength(workernode->thread_state[my_id].bg, L);
          p7_bg_NullOne(workernode->thread_state[my_id].bg, (const ESL_DSQ *) the_sequence, L, &nullsc);
          p7_SSVFilter((const ESL_DSQ *) the_sequence, L, workernode->thread_state[my_id].om, &retsc);
          cuda_config->card_mem[my_id].cpu_hits[my_stream][num_sequences] = (retsc - nullsc) / eslCONST_LOG2; */
          // uncomment this code to fill cpu_hits with ssv scores so we can check filter results in CUDA land
    current_offset += L_effective;
    the_sequence += L + 2;
    sequence_start = the_sequence;
    seq_id = *((uint64_t *) the_sequence);
    the_sequence += sizeof(uint64_t);
    L = *((uint64_t *) the_sequence);
    the_sequence += sizeof(uint64_t);
    num_sequences += 1;
    L_effective = (L + 31) & ALIGNEIGHT_MASK;
    //printf("num_sequences = %d, seq_id = %lu, current_offset = %lu, last_sequence = %lu, work_end = %lu\n", num_sequences, seq_id, current_offset, last_sequence, work_end);
  }
#ifdef __NVVP_PROFILE_ON
  nvtxRangeEnd(prep);
#endif
  //printf("GPU started chunk with %d sequences\n", num_sequences);
  cuda_config->card_mem[my_id].num_sequences[stream] = num_sequences;
  // Copy the input data to the card
  // uncomment this to check SSV p7_cuda_wrapper(hipMemcpy(cuda_config->card_mem[my_id].gpu_hits[my_stream], cuda_config->card_mem[my_id].cpu_hits[my_stream], num_sequences * sizeof(uint64_t), hipMemcpyHostToDevice));
#ifdef __NVVP_PROFILE_ON
  sprintf(send_event_string, "Send %d", stream);  // This will fail miserably if stream is more than 10 digits, so don't do that
  nvtxRangeId_t send = nvtxRangeStartEx(&send_event);
#endif
  p7_cuda_wrapper(hipMemcpyAsync(cuda_config->card_mem[my_id].gpu_data[stream], cuda_config->card_mem[my_id].cpu_data[stream], current_offset, hipMemcpyHostToDevice, cuda_config->card_mem[my_id].streams[stream]));
  //printf("GPU sequences range from addresses %p to %p\n", cuda_config->card_mem[my_id].gpu_data[my_stream], cuda_config->card_mem[my_id].gpu_data[my_stream] + (current_offset -1));
  p7_cuda_wrapper(hipMemcpyAsync(cuda_config->card_mem[my_id].gpu_offsets[stream], cuda_config->card_mem[my_id].cpu_offsets[stream], num_sequences *sizeof(uint64_t), hipMemcpyHostToDevice, cuda_config->card_mem[my_id].streams[stream]));

  p7_cuda_wrapper(hipMemcpyAsync(cuda_config->card_mem[my_id].gpu_lengths[stream], cuda_config->card_mem[my_id].cpu_lengths[stream], num_sequences *sizeof(uint64_t), hipMemcpyHostToDevice, cuda_config->card_mem[my_id].streams[stream]));
#ifdef __NVVP_PROFILE_ON
  nvtxRangeEnd(send);
#endif
#ifdef __NVVP_PROFILE_ON
  sprintf(comp_event_string, "Comp %d", stream);  // This will fail miserably if stream is more than 10 digits, so don't do that
  nvtxRangeId_t comp = nvtxRangeStartEx(&comp_event);
#endif
  //printf("GPU Thread %d starting chunk with %d sequences and length %lu\n", my_id, num_sequences, current_offset);
  p7_orion<<<num_blocks, threads_per_block, 0, cuda_config->card_mem[my_id].streams[stream]>>>(num_sequences, (uint8_t *) cuda_config->card_mem[my_id].gpu_data[stream], cuda_config->card_mem[my_id].gpu_lengths[stream], cuda_config->card_mem[my_id].gpu_offsets[stream], cuda_config->card_mem[my_id].gpu_hits[stream], cuda_config->card_mem[my_id].gpu_scores[stream], om, mu, lambda);
  p7_kernel_error_check();
#ifdef __NVVP_PROFILE_ON
  nvtxRangeEnd(comp);
#endif
#ifdef __NVVP_PROFILE_ON
  sprintf(recv_event_string, "Recv %d", stream);  // This will fail miserably if stream is more than 10 digits, so don't do that
  nvtxRangeId_t recv = nvtxRangeStartEx(&recv_event);
#endif
  // Get the results back
  p7_cuda_wrapper(hipMemcpyAsync(cuda_config->card_mem[my_id].cpu_hits[stream], cuda_config->card_mem[my_id].gpu_hits[stream], num_sequences *sizeof(int8_t) ,hipMemcpyDeviceToHost, cuda_config->card_mem[my_id].streams[stream]));
  p7_cuda_wrapper(hipMemcpyAsync(cuda_config->card_mem[my_id].cpu_scores[stream], cuda_config->card_mem[my_id].gpu_scores[stream], num_sequences *sizeof(float) ,hipMemcpyDeviceToHost, cuda_config->card_mem[my_id].streams[stream]));
#ifdef __NVVP_PROFILE_ON
  nvtxRangeEnd(recv);
#endif
  // update values for next call
  *chunk_end = seq_id;
  *sequence_data = the_sequence - (2* sizeof(uint64_t)); //subtract off the two uint64_t we looked at to determine that the sequence wouldn't fit

  return;
}

void parse_CUDA_chunk_results(P7_DAEMON_WORKERNODE_STATE *workernode, P7_CUDA_CONFIG *cuda_config, uint32_t my_id, uint32_t stream){
#ifdef __NVVP_PROFILE_ON
  char wait_event_string[] = "Wait -1111111111";
  nvtxEventAttributes_t wait_event = {0};
  wait_event.version = NVTX_VERSION;
  wait_event.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  wait_event.colorType = NVTX_COLOR_ARGB;
  wait_event.color = 0xFF0000; //red
  wait_event.messageType = NVTX_MESSAGE_TYPE_ASCII;
  wait_event.message.ascii = wait_event_string;

  char pass_event_string[] = "Pass -1111111111";
  nvtxEventAttributes_t pass_event = {0};
  pass_event.version = NVTX_VERSION;
  pass_event.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  pass_event.colorType = NVTX_COLOR_ARGB;
  pass_event.color = 0xFF00BB; //purple
  pass_event.messageType = NVTX_MESSAGE_TYPE_ASCII;
  pass_event.message.ascii = pass_event_string;
#endif 

#ifdef __NVVP_PROFILE_ON
  sprintf(wait_event_string, "Wait %d", stream);  // This will fail miserably if stream is more than 10 digits, so don't do that
  nvtxRangeId_t waitev = nvtxRangeStartEx(&wait_event);
#endif
    // First, synchronize so that we're sure the stream is done computing and copying data
  hipStreamSynchronize(cuda_config->card_mem[my_id].streams[stream]);
#ifdef __NVVP_PROFILE_ON
  nvtxRangeEnd(waitev);
#endif  
#ifdef __NVVP_PROFILE_ON
  sprintf(pass_event_string, "Pass %d", stream);  // This will fail miserably if stream is more than 10 digits, so don't do that
  nvtxRangeId_t passev = nvtxRangeStartEx(&pass_event);
#endif
  int num_hits = 0;
  P7_BACKEND_QUEUE_ENTRY *chain_start = NULL;
  P7_BACKEND_QUEUE_ENTRY *chain_end = NULL;
  for(int q = 0; q < cuda_config->card_mem[my_id].num_sequences[stream]; q++){

    if(cuda_config->card_mem[my_id].cpu_hits[stream][q] !=0){ //This sequence hit
      // get an entry to put this comparison in
      P7_BACKEND_QUEUE_ENTRY * the_entry = workernode_get_backend_queue_entry_from_pool(workernode);
      the_entry->seq_position = q;
      the_entry->seq_in_chunk = cuda_config->card_mem[my_id].num_sequences[stream];
      // Skip the sparsemask swapping, as the GPU doesn't do enough of the overthruster to
      // populate a sparse mask
      the_entry->score = cuda_config->card_mem[my_id].cpu_scores[stream][q];
      // populate the fields
      char *s = cuda_config->card_mem[my_id].cpu_sequences[stream][q];
      the_entry->seq_id = *((uint64_t *) s);
      the_entry->sequence = (ESL_DSQ *) (s+ 2*sizeof(uint64_t));
      the_entry->L = cuda_config->card_mem[my_id].cpu_lengths[stream][q];
      the_entry->do_overthruster = 1;
      the_entry->next = NULL;
      workernode->thread_state[my_id].comparisons_queued += 1;
      // put the entry in the chain
      the_entry->next = chain_start; // put new entries on the front
      chain_start = the_entry;
      if (chain_end ==NULL){
        chain_end = the_entry;
      }
      num_hits++;
    }
  }
  if(num_hits > 0){
    workernode_put_backend_chain_in_queue(workernode, num_hits, chain_start, chain_end);
  }
  #ifdef __NVVP_PROFILE_ON
  nvtxRangeEnd(passev);
#endif 
}